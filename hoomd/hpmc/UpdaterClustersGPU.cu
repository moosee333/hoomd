
#include <hip/hip_runtime.h>
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#ifdef NVGRAPH_AVAILABLE

#include "UpdaterClustersGPU.cuh"

#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>

#include <queue>

#include <cublas_v2.h>
#include <nvgraph.h>

/*! \file UpdaterClustersGPU.cu
    \brief Implements a spectral method for finding strongly connected components
*/

namespace hpmc
{

namespace detail
{

#define check_nvgraph(a) \
    {\
    nvgraphStatus_t status = (a);\
    if ((int)status != NVGRAPH_STATUS_SUCCESS)\
        {\
        printf("nvgraph ERROR %d in file %s line %d\n",status,__FILE__,__LINE__);\
        throw std::runtime_error("Error during clusters update");\
        }\
    }

#define check_cublas(a) \
    {\
    cublasStatus_t status = (a);\
    if ((int)status != CUBLAS_STATUS_SUCCESS)\
        {\
        printf("cublas ERROR %d in file %s line %d\n",status,__FILE__,__LINE__);\
        throw std::runtime_error("Error during clusters update");\
        }\
    }

#define check_cuda(a) \
    {\
    cudaError_t status = (a);\
    if ((int)status != cudaSuccess)\
        {\
        return status;\
        }\
    }


struct get_source : public thrust::unary_function<uint2, unsigned int>
    {
    __host__ __device__
    unsigned int operator()(const uint2& u) const
        {
        return u.x;
        }
    };

struct get_destination : public thrust::unary_function<uint2, unsigned int>
    {
    __host__ __device__
    unsigned int operator()(const uint2& u) const
        {
        return u.y;
        }
    };

struct greater_equal_x : public thrust::unary_function<float, bool>
    {
    __host__ __device__
    greater_equal_x(const float _x)
        : x(_x)
        { }

    __host__ __device__
    bool operator()(const float& f) const
        {
        return f >= x;
        }

    float x;
    };

struct my_int_as_float : public thrust::unary_function<int, float>
    {
    __device__
    float operator()(const int& i) const
        {
        return __int_as_float(i);
        }
    };

struct my_float_as_int : public thrust::unary_function<float, int>
    {
    __device__
    int operator()(const float& f) const
        {
        return __float_as_int(f);
        }
    };

struct is_reachable : public thrust::unary_function<int, bool>
    {
    __host__ __device__
    bool operator()(const int& i) const
        {
        return i != 2147483647; // 2^31-1
        }
    };

cudaError_t gpu_connected_components(
    const uint2 *d_adj,
    unsigned int N,
    unsigned int n_elements,
    unsigned int *d_components,
    unsigned int &num_components,
    cudaStream_t stream,
    unsigned int max_ites,
    float tol,
    float jump_tol,
    const CachedAllocator& alloc)
    {
    // input matrix in COO format
    nvgraphCOOTopology32I_t COO_input;
    COO_input = (nvgraphCOOTopology32I_t) malloc(sizeof(struct nvgraphCOOTopology32I_st));

    COO_input->nvertices = N;
    COO_input->nedges = 2*n_elements;  // for undirected graph
    COO_input->tag = NVGRAPH_UNSORTED;

    float *d_edge_data_coo;
    float *d_edge_data_csr;

    // allocate COO matrix topology
    check_cuda(cudaMalloc((void **)&(COO_input->source_indices), COO_input->nedges*sizeof(int)));
    check_cuda(cudaMalloc((void **)&(COO_input->destination_indices), COO_input->nedges*sizeof(int)));

    // fill sparse matrix and make it symmetric
    thrust::device_ptr<const uint2> adj(d_adj);
    auto source = thrust::make_transform_iterator(adj, get_source());
    auto destination = thrust::make_transform_iterator(adj, get_destination());
    thrust::device_ptr<int> coo_source(COO_input->source_indices);
    thrust::device_ptr<int> coo_destination(COO_input->destination_indices);

    thrust::copy(source, source+n_elements, coo_source);
    thrust::copy(destination, destination+n_elements, coo_destination);

    // transpose
    thrust::copy(source, source+n_elements, coo_destination+n_elements);
    thrust::copy(destination, destination+n_elements, coo_source+n_elements);

    nvgraphCSRTopology32I_t CSR_output;
    CSR_output = (nvgraphCSRTopology32I_t) malloc(sizeof(struct nvgraphCSRTopology32I_st));

    // allocate CSR matrix topology
    check_cuda(cudaMalloc((void **)&(CSR_output->source_offsets), (COO_input->nvertices+1)*sizeof(int)));
    check_cuda(cudaMalloc((void **)&(CSR_output->destination_indices), COO_input->nedges*sizeof(int)));

    // allocate edge data
    check_cuda(cudaMalloc((void **)&d_edge_data_coo, COO_input->nedges*sizeof(float)));
    check_cuda(cudaMalloc((void **)&d_edge_data_csr, COO_input->nedges*sizeof(float)));

    // put ones on the elements of the adjacency matrix in COO format
    thrust::device_ptr<float> edge_data_coo(d_edge_data_coo);
    thrust::fill(thrust::cuda::par(alloc), edge_data_coo, edge_data_coo + COO_input->nedges, 1.0);

    // create nvgraph handle
    nvgraphHandle_t nvgraphH;
    check_nvgraph(nvgraphCreate(&nvgraphH));

    // create parent graph object
    nvgraphGraphDescr_t graph;
    check_nvgraph(nvgraphCreateGraphDescr(nvgraphH, &graph));

    // convert COO->CSR
    cudaDataType_t edge_dimT = CUDA_R_32F;
    check_nvgraph(nvgraphConvertTopology(
        nvgraphH,
        NVGRAPH_COO_32,
        COO_input,
        d_edge_data_coo,
        &edge_dimT,
        NVGRAPH_CSR_32,
        CSR_output,
        d_edge_data_csr));

    // these variables will track the dimensions of the current subgraph
    unsigned int nverts = CSR_output->nvertices;

    // set graph connectivity and properties
    unsigned int edge_num_sets = 1;

    check_nvgraph(nvgraphSetGraphStructure(
        nvgraphH,
        graph,
        (void *) CSR_output,
        NVGRAPH_CSR_32));
    check_nvgraph(nvgraphAllocateEdgeData(
        nvgraphH,
        graph,
        edge_num_sets,
        &edge_dimT));
    check_nvgraph(nvgraphSetEdgeData(
        nvgraphH,
        graph,
        d_edge_data_csr,
        0 // edge data set 0
        ));

    /* Vertex data
     * set 0: vector to multiply by
     * set 1: vector to add, and output
     * set 2: origin particle index for connected component
     * set 3: distances from BFS traversal
     */
    int vertex_num_sets = 4;
    cudaDataType_t vertex_dimT[vertex_num_sets];
    for (int i = 0; i < vertex_num_sets; ++i)
        vertex_dimT[i] = CUDA_R_32F;

    check_nvgraph(nvgraphAllocateVertexData(
        nvgraphH,
        graph,
        vertex_num_sets,
        vertex_dimT));

    // set up cublas handle
    cublasHandle_t cublasH = NULL;

    check_cublas(cublasCreate(&cublasH));
    check_cublas(cublasSetStream(cublasH, stream));

    // a RHS vector of ones
    float *d_ones_float;
    check_cuda(cudaMalloc((void **)&d_ones_float, nverts*sizeof(float)));
    thrust::device_ptr<float> ones_float(d_ones_float);
    thrust::fill(ones_float, ones_float + nverts, 1.0);

    // stores the diagonal matrix
    float *d_diag;
    check_cuda(cudaMalloc((void **)&d_diag, nverts*sizeof(float)));

    // solution vector
    float *d_x;
    check_cuda(cudaMalloc((void **) &d_x, nverts*sizeof(float)));
    size_t x_index = 0; // vertex data 0

    // sorted vertex indices for subgraph
    int *d_vertices;
    check_cuda(cudaMalloc((void **)&d_vertices, nverts*sizeof(int)));
    thrust::device_ptr<int> vertices(d_vertices);

    // LHS of matrix vector multiplication y, and additive input vector
    float *d_y;
    check_cuda(cudaMalloc((void **)&d_y, nverts*sizeof(float)));

    thrust::device_ptr<float> y(d_y);
    thrust::device_ptr<float> diag(d_diag);
    thrust::device_ptr<float> x(d_x);

    // y is vertex data 1
    size_t y_index = 1;

    // attach the ascending particle index as vertex data 2
    float *d_ptl_idx;
    check_cuda(cudaMalloc((void **)&d_ptl_idx, nverts*sizeof(float)));
    thrust::device_ptr<float> ptl_idx(d_ptl_idx);
    auto ptl_idx_as_float = thrust::make_transform_iterator(
        thrust::counting_iterator<int>(0),
        my_int_as_float());
    thrust::copy(
        thrust::cuda::par(alloc),
        ptl_idx_as_float,
        ptl_idx_as_float + N,
        ptl_idx);

    size_t ptls_index = 2; // vertex set 2
    check_nvgraph(nvgraphSetVertexData(
        nvgraphH,
        graph,
        d_ptl_idx,
        ptls_index
        ));

    // difference between x shifted by one and x tself
    float *d_delta_x;
    check_cuda(cudaMalloc((void **) &d_delta_x, sizeof(float)*nverts));
    thrust::device_ptr<float> delta_x(d_delta_x);

    // traversal distances as graph vertex data 3
    int *d_distances;
    check_cuda(cudaMalloc((void **) &d_distances, sizeof(int)*nverts));

    thrust::device_ptr<int> distances(d_distances);
    size_t distances_index = 3;

    // one component
    float *d_component;
    check_cuda(cudaMalloc((void **) &d_component, sizeof(float)*nverts));
    thrust::device_ptr<float> component(d_component);

    // wrap the components output vector
    thrust::device_ptr<unsigned int> components(d_components);

    // a queue for subgraphs (BFS over components)
    std::queue<nvgraphGraphDescr_t> Q;

    // push the parent graph
    Q.push(graph);

    num_components = 0;

    // iteratively partition the graph until all connected components are found
    while (!Q.empty())
        {
        // pop the graph handle from the top of the queue
        auto cur_graph = Q.front();
        Q.pop();

        // get current number of vertices and edges
        struct nvgraphCSRTopology32I_st cur_topology;
        cur_topology.source_offsets = NULL;
        cur_topology.destination_indices = NULL;

        check_nvgraph(nvgraphGetGraphStructure(
            nvgraphH,
            cur_graph,
            &cur_topology,
            NULL));

        nverts = cur_topology.nvertices;

        bool done = false;

        if (nverts == 1)
            {
            // we found a single disconnected vertex, skip spectral analysis
            done = true;
            }
        else
            {
            // use eigenvalue decomposition and estimate partitions from discontinuous steps in the eigenvector associated
            // with the smallest eigenvalue of the Laplacian

            printf("> %d %d\n", nverts, cur_topology.nedges);

            /*
             * compute Laplacian L = diag(A.e) - A
             */
            // LHS y = -diag(A.e)
            float h_one = 1.0;
            float h_minusone = -1.0;
            float zero = 0.0;

            // set vertex data for x = e
            check_nvgraph(nvgraphSetVertexData(
                nvgraphH,
                cur_graph,
                d_ones_float,
                x_index
                ));

            check_nvgraph(nvgraphSrSpmv(
                nvgraphH,
                cur_graph,
                0, // edge set
                &h_minusone,
                x_index, // vertex set for multiplication
                &zero, // multiplying value
                y_index, // vertex set for output
                NVGRAPH_PLUS_TIMES_SR));

            // extract result from matrix vector multiplication into d_diag
            check_nvgraph(nvgraphGetVertexData(
                nvgraphH,
                cur_graph,
                (void *) d_diag,
                y_index));

            // -L = A - diag(A.e)
            // since sparse matrix addition (subtraction) is cumbersome we will carry A and -diag(A.e) separately

            /* find the largest eigenvalue of the negative semidefinite matrix -L (== the singular value of L corresponding
               to a connected component) using the power method

                http://docs.nvidia.com/cuda/cusparse/index.html#csrmv_examples
             */

            float lambda = 0.0;
            float lambda_next = 0.0;

            // initial guess x0 = ones
            check_cuda(cudaMemcpy(d_x, d_ones_float, sizeof(float) * nverts, cudaMemcpyDeviceToDevice));

            for (unsigned int ite = 0; ite < max_ites; ite++)
                {
                /* normalize vector
                 * x= x/|x|
                 */
                float nrm2_x;
                check_cublas(cublasSnrm2_v2(cublasH,
                    nverts,
                    d_x,
                    1, // incx
                    &nrm2_x
                    ));

                float one_over_nrm2_x = 1.0 / nrm2_x;
                check_cublas(cublasSscal_v2(cublasH,
                    nverts,
                    &one_over_nrm2_x,
                    d_x,
                    1 // incx
                    ));

                    {
                    printf("==== ITERATION %d\n", ite);
                    printf("norm %f\n", nrm2_x);
                    }

                /*
                 * y = -L*x = A*x - diag(A.e)*x
                 */

                // y = diag(A.e)*x (component wise multiplication)
                thrust::transform(
                    x,
                    x + nverts,
                    diag,
                    y,
                    thrust::multiplies<float>()
                    );

                // update vertex data for x
                check_nvgraph(nvgraphSetVertexData(
                    nvgraphH,
                    cur_graph,
                    d_x,
                    x_index
                    ));

                // update vertex data for y
                check_nvgraph(nvgraphSetVertexData(
                    nvgraphH,
                    cur_graph,
                    d_y,
                    y_index
                    ));

                // y=A*x + y
                check_nvgraph(nvgraphSrSpmv(
                    nvgraphH,
                    cur_graph,
                    0, // edge set
                    &h_one,
                    x_index, // vertex set for multiplication
                    &h_one,
                    y_index, // vertex set for addition
                    NVGRAPH_PLUS_TIMES_SR));

                // extract vertex data from graph
                check_nvgraph(nvgraphGetVertexData(
                    nvgraphH,
                    cur_graph,
                    (void *) d_y,
                    y_index));

                /*
                 * lambda = y**T*x
                 */
                check_cublas(cublasSdot_v2(
                    cublasH,
                    nverts,
                    d_x,
                    1, // incx,
                    d_y,
                    1, // incy
                    &lambda_next
                    ));

                /*
                 * check if convergence
                 */
                if ( (ite > 0) && fabs(lambda - lambda_next) < tol)
                    break;

                /*
                 * x := y
                 * lambda = lambda_next
                 */
                check_cuda(cudaMemcpy(d_x, d_y, nverts*sizeof(float), cudaMemcpyDeviceToDevice));
                lambda = lambda_next;
                } // end of an iteration of the power method

            /*
             * x contains eigenvector corresponding to the largest eigenvalue lambda of -L
             */

            // recover the eigenvalues of the positive semidefinite matrix L by transforming in place x-> -x
            thrust::transform(
                thrust::cuda::par(alloc),
                x,
                x + nverts,
                x,
                thrust::negate<float>());

            // fill vertices with ascending sequence
            auto count = thrust::counting_iterator<int>(0);
            thrust::copy(
                thrust::cuda::par(alloc),
                count,
                count + nverts,
                vertices);

            // sort vertex indices by key (ith component of eigenvector)
            thrust::sort_by_key(
                thrust::cuda::par(alloc),
                x,
                x + nverts,
                vertices);

            // determine a jump in x by taking the adjacent difference
            thrust::transform(
                thrust::cuda::par(alloc),
                x + 1,
                x + nverts,
                x,
                delta_x,
                thrust::minus<float>()
                );

            // pick up the jump at i
            auto jump_it = thrust::find_if(
                thrust::cuda::par(alloc),
                delta_x,
                delta_x + nverts - 1,
                greater_equal_x(jump_tol)
                );

            // last index of subgraph + 1
            unsigned int split_idx = (jump_it - delta_x) + 1;

            if (split_idx == nverts)
                {
                // we found a candidate for a connected component, check if it cannot be split further
                printf("%d %d\n", nverts, cur_topology.nedges);
                nvgraphTraversalParameter_t traversal_param;
                nvgraphTraversalParameterInit(&traversal_param);
                nvgraphTraversalSetDistancesIndex(&traversal_param, distances_index);
                nvgraphTraversalSetUndirectedFlag(&traversal_param, true);

                // do a BFS traversal starting from first index of this putative s.c. component
                int source_vert = 0;
                check_nvgraph(nvgraphTraversal(
                    nvgraphH,
                    cur_graph,
                    NVGRAPH_TRAVERSAL_BFS,
                    &source_vert,
                    traversal_param));

                // extract the distances
                check_nvgraph(nvgraphGetVertexData(
                    nvgraphH,
                    cur_graph,
                    (void *)d_distances,
                    distances_index
                    ));

                /*
                 * if any index is unreachable (distance == 2^31 - 1), this component can still be split
                 */

                // reset vertices to ascending sequence
                thrust::copy(
                    thrust::cuda::par(alloc),
                    count,
                    count + nverts,
                    vertices);

                // sort vertex indices by distance from source vertex
                thrust::sort_by_key(
                    thrust::cuda::par(alloc),
                    distances,
                    distances + nverts,
                    vertices);

                // find first unreachable vertex
                auto unreachable_it = thrust::partition_point(
                    thrust::cuda::par(alloc),
                    distances,
                    distances + nverts,
                    is_reachable()
                    );
                split_idx = unreachable_it - distances;

                if (split_idx == nverts)
                    {
                    // we have found a strongly connected component
                    done = true;
                    }
                } // end if we found a s.c. component candidate

            if (! done)
                {
                // sort the indices to the left of split_idx as required by nvgraph
                thrust::sort(
                    thrust::cuda::par(alloc),
                    vertices,
                    vertices + split_idx
                    );

                // create subgraph object to the left
                nvgraphGraphDescr_t sub_graph_left;
                check_nvgraph(nvgraphCreateGraphDescr(nvgraphH, &sub_graph_left));

                // extract the subgraph
                check_nvgraph(nvgraphExtractSubgraphByVertex(
                    nvgraphH,
                    cur_graph,
                    sub_graph_left,
                    d_vertices,
                    split_idx));

                // push the left subgraph in the queue
                Q.push(sub_graph_left);

                // sort the indices to the right of split_idx
                thrust::sort(
                    thrust::cuda::par(alloc),
                    vertices + split_idx,
                    vertices + nverts
                    );

                // create subgraph object to the right
                nvgraphGraphDescr_t sub_graph_right;
                check_nvgraph(nvgraphCreateGraphDescr(nvgraphH, &sub_graph_right));

                // extract the subgraph
                check_nvgraph(nvgraphExtractSubgraphByVertex(
                    nvgraphH,
                    cur_graph,
                    sub_graph_right,
                    d_vertices+split_idx,
                    nverts - split_idx));

                // push the right subgraph in the queue
                Q.push(sub_graph_right);
                }
            } // end if finite connected component

        if (done)
            {
            // extract the particle indices of the connected component
            check_nvgraph(nvgraphGetVertexData(
                nvgraphH,
                cur_graph,
                (void *) d_component,
                ptls_index));

            printf("%p %d\n", d_components, nverts);

            // label the particle indices in the output array by the index of this connected component

            // scatter the component indices as obtained from the vertex data
            auto component_idx = thrust::make_transform_iterator(
                component,
                my_float_as_int());

            auto scatter_it = thrust::make_permutation_iterator(
                components,  // the output vector
                component_idx    // the indices vector
                );

                {
                int *d_indices;
                check_cuda(cudaMalloc(&d_indices, sizeof(int)*nverts));
                thrust::device_ptr<int> indices(d_indices);
                thrust::copy(component_idx, component_idx+ nverts, indices);
                int h_indices[nverts];
                check_cuda(cudaMemcpy(h_indices, d_indices, nverts*sizeof(int),cudaMemcpyDeviceToHost));
                for (unsigned int i = 0; i < nverts; ++i)
                    {
                    int j = h_indices[i];
                    printf("%d\n", j);
                    }
                cudaFree(d_indices);
                }

            thrust::fill(
                thrust::cuda::par(alloc),
                scatter_it,
                scatter_it + nverts,
                num_components++);
            }


        // release this graph descriptor
        check_nvgraph(nvgraphDestroyGraphDescr(nvgraphH, cur_graph));
        };

    // free device data
    check_cuda(cudaFree(d_component));
    check_cuda(cudaFree(d_distances));
    check_cuda(cudaFree(d_ptl_idx));
    check_cuda(cudaFree(d_vertices));
    check_cuda(cudaFree(d_delta_x));
    check_cuda(cudaFree(d_y));
    check_cuda(cudaFree(d_x));
    check_cuda(cudaFree(d_diag));
    check_cuda(cudaFree(d_ones_float));

    check_cuda(cudaFree(COO_input->source_indices));
    check_cuda(cudaFree(COO_input->destination_indices));
    check_cuda(cudaFree(CSR_output->source_offsets));
    check_cuda(cudaFree(CSR_output->destination_indices));
    check_cuda(cudaFree(d_edge_data_csr));
    check_cuda(cudaFree(d_edge_data_coo));

    // release nvgraph handle
    if (nvgraphH)
        check_nvgraph(nvgraphDestroy(nvgraphH));

    free(COO_input);
    free(CSR_output);

    // clean cublas
    if (cublasH)
        cublasDestroy(cublasH);

    return cudaSuccess;
    }

} // end namespace detail
} // end namespace hpmc
#endif // NVGRAPH_AVAILABLE
