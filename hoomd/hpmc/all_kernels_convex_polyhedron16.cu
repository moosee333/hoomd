// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"

#include "ShapeSphere.h"
#include "ShapeConvexPolygon.h"
#include "ShapePolyhedron.h"
#include "ShapeConvexPolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSpheropolygon.h"
#include "ShapeSimplePolygon.h"
#include "ShapeEllipsoid.h"
#include "ShapeFacetedSphere.h"
#include "ShapeSphinx.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeConvexPolyhedron<16>
template hipError_t gpu_hpmc_free_volume<ShapeConvexPolyhedron<16> >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeConvexPolyhedron<16> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<16> >(const hpmc_args_t& args,
                                                  const typename ShapeConvexPolyhedron<16> ::param_type *d_params);
template void gpu_hpmc_implicit_count_overlaps<ShapeConvexPolyhedron<16> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron<16> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeConvexPolyhedron<16> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron<16> ::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
