// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"

#include "ShapeSphere.h"
#include "ShapeConvexPolygon.h"
#include "ShapePolyhedron.h"
#include "ShapeConvexPolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSpheropolygon.h"
#include "ShapeSimplePolygon.h"
#include "ShapeEllipsoid.h"
#include "ShapeFacetedSphere.h"
#include "ShapeSphinx.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeUnion<ShapeSphere, 16>
template hipError_t gpu_hpmc_free_volume<ShapeUnion<ShapeSphere, 16> >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeUnion<ShapeSphere, 16> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeUnion<ShapeSphere, 16> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere, 16> ::param_type *d_params);
template void gpu_hpmc_implicit_count_overlaps<ShapeUnion<ShapeSphere, 16> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere, 16> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeUnion<ShapeSphere, 16> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere, 16> ::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
