// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "UpdaterMuVTGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"
#include "IntegratorHPMCMonoImplicitNewGPU.cuh"

#include "ShapeSpheropolyhedron.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeSpheropolyhedron
template hipError_t gpu_hpmc_free_volume<ShapeSpheropolyhedron >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_update_aabb<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_moves<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_check_overlaps<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_accept<ShapeSpheropolyhedron >(const hpmc_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_count_overlaps<ShapeSpheropolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeSpheropolyhedron >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_queue<ShapeSpheropolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_insert_depletants_dp<ShapeSpheropolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject_new<ShapeSpheropolyhedron >(const hpmc_implicit_args_new_t& args,
                                                  const typename ShapeSpheropolyhedron ::param_type *d_params);
template hipError_t gpu_hpmc_muvt<ShapeSpheropolyhedron >(const hpmc_muvt_args_t &args,
                                                       const typename ShapeSpheropolyhedron ::param_type *d_params);
}; // end namespace detail

} // end namespace hpmc
