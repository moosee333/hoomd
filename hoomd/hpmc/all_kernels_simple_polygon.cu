// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"

#include "ShapeSphere.h"
#include "ShapeConvexPolygon.h"
#include "ShapePolyhedron.h"
#include "ShapeConvexPolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSpheropolygon.h"
#include "ShapeSimplePolygon.h"
#include "ShapeEllipsoid.h"
#include "ShapeFacetedSphere.h"
#include "ShapeSphinx.h"
#include "ShapeUnion.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeSimplePolygon
template hipError_t gpu_hpmc_free_volume<ShapeSimplePolygon>(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSimplePolygon>(const hpmc_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template void gpu_hpmc_implicit_count_overlaps<ShapeSimplePolygon>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeSimplePolygon>(const hpmc_implicit_args_t& args,
                                                  const typename ShapeSimplePolygon::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
