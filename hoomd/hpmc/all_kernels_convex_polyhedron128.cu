// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "ComputeFreeVolumeGPU.cuh"
#include "IntegratorHPMCMonoGPU.cuh"
#include "IntegratorHPMCMonoImplicitGPU.cuh"

#include "ShapeConvexPolyhedron.h"

namespace hpmc
{

namespace detail
{

//! HPMC kernels for ShapeConvexPolyhedron<128>
template hipError_t gpu_hpmc_free_volume<ShapeConvexPolyhedron<128> >(const hpmc_free_volume_args_t &args,
                                                       const typename ShapeConvexPolyhedron<128> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<128> >(const hpmc_args_t& args,
                                                  const typename ShapeConvexPolyhedron<128> ::param_type *d_params);
template void gpu_hpmc_implicit_count_overlaps<ShapeConvexPolyhedron<128> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron<128> ::param_type *d_params);
template hipError_t gpu_hpmc_implicit_accept_reject<ShapeConvexPolyhedron<128> >(const hpmc_implicit_args_t& args,
                                                  const typename ShapeConvexPolyhedron<128> ::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
