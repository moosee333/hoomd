#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: mphoward

#include "NeighborListGPUStencil.cuh"
#include "hoomd/TextureTools.h"
#include "hoomd/extern/cub/hipcub/hipcub.hpp"

/*! \file NeighborListGPUStencil.cu
    \brief Defines GPU kernel code for O(N) neighbor list generation on the GPU with multiple bin stencils
*/

//! Texture for reading d_cell_xyzf
scalar4_tex_t cell_xyzf_1d_tex;

//! Texture for reading d_cell_tdb
scalar4_tex_t cell_tdb_1d_tex;

//! Texture for reading d_stencil
scalar4_tex_t stencil_1d_tex;

//! Warp-centric scan (Kepler and later)
template<int NT>
struct warp_scan_sm30_stencil
    {
    __device__ static int Scan(int tid, unsigned char x, unsigned char* total)
        {
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));

        int first = laneid - tid;

        #pragma unroll
        for(int offset = 1; offset < NT; offset += offset)
            {
            int y = __shfl(x,(first + tid - offset) &(WARP_SIZE -1));
            if(tid >= offset) x += y;
            }

        // all threads get the total from the last thread in the cta
        *total = __shfl(x,first + NT - 1);

        // shift by one (exclusive scan)
        int y = __shfl(x,(first + tid - 1) &(WARP_SIZE-1));
        x = tid ? y : 0;

        return x;
        }
    };

//! Kernel call for generating neighbor list on the GPU using multiple stencils (Kepler optimized version)
/*! \tparam flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering.
    \tparam threads_per_particle Number of threads cooperatively computing the neighbor list
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param d_Nmax Maximum number of neighbors per type
    \param d_head_list List of indexes to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param d_cell_size Number of particles in each cell
    \param d_cell_xyzf Cell contents (xyzf array from CellList with flag=type)
    \param d_cell_tdb Cell contents (tdb array from CellList with)
    \param ci Cell indexer for indexing cells
    \param cli Cell list indexer for indexing into d_cell_xyzf
    \param d_stencil 2D array of stencil offsets per type
    \param d_n_stencil Number of stencils per type
    \param stencil_idx Indexer into \a d_stencil
    \param box Simulation box dimensions
    \param d_r_cut Cutoff radius stored by pair type r_cut(i,j)
    \param r_buff The maximum radius for which to include particles as neighbors
    \param ntypes Number of particle types
    \param ghost_width Width of ghost cell layer

    \note optimized for Kepler
*/
template<unsigned char flags, int threads_per_particle>
__global__ void gpu_compute_nlist_stencil_kernel(unsigned int *d_nlist,
                                                 unsigned int *d_n_neigh,
                                                 Scalar4 *d_last_updated_pos,
                                                 unsigned int *d_conditions,
                                                 const unsigned int *d_Nmax,
                                                 const unsigned int *d_head_list,
                                                 const unsigned int *d_pid_map,
                                                 const Scalar4 *d_pos,
                                                 const unsigned int *d_body,
                                                 const Scalar *d_diameter,
                                                 const unsigned int N,
                                                 const unsigned int *d_cell_size,
                                                 const Scalar4 *d_cell_xyzf,
                                                 const Scalar4 *d_cell_tdb,
                                                 const Index3D ci,
                                                 const Index2D cli,
                                                 const Scalar4 *d_stencil,
                                                 const unsigned int *d_n_stencil,
                                                 const Index2D stencil_idx,
                                                 const BoxDim box,
                                                 const Scalar *d_r_cut,
                                                 const Scalar r_buff,
                                                 const unsigned int ntypes,
                                                 const Scalar3 ghost_width)
    {
    bool filter_body = flags & 1;
    bool diameter_shift = flags & 2;

    // cache the r_listsq parameters into shared memory
    Index2D typpair_idx(ntypes);
    const unsigned int num_typ_parameters = typpair_idx.getNumElements();

    // shared data for per type pair parameters
    extern __shared__ unsigned char s_data[];

    // pointer for the r_listsq data
    Scalar *s_r_list = (Scalar *)(&s_data[0]);
    unsigned int *s_Nmax = (unsigned int *)(&s_data[sizeof(Scalar)*num_typ_parameters]);

    // load in the per type pair r_list
    for (unsigned int cur_offset = 0; cur_offset < num_typ_parameters; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < num_typ_parameters)
            {
            Scalar r_cut = d_r_cut[cur_offset + threadIdx.x];
            // force the r_list(i,j) to a skippable value if r_cut(i,j) is skippable
            s_r_list[cur_offset + threadIdx.x] = (r_cut > Scalar(0.0)) ? r_cut+r_buff : Scalar(-1.0);
            }
        if (cur_offset + threadIdx.x < ntypes)
            {
            s_Nmax[cur_offset + threadIdx.x] = d_Nmax[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    // each set of threads_per_particle threads is going to compute the neighbor list for a single particle
    int idx;
    if (gridDim.y > 1)
        {
        // fermi workaround
        idx = (blockIdx.x + blockIdx.y*65535) * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }
    else
        {
        idx = blockIdx.x * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }

    // one thread per particle
    if (idx >= N) return;

    // get the write particle id
    int my_pidx = d_pid_map[idx];

    Scalar4 my_postype = d_pos[my_pidx];
    Scalar3 my_pos = make_scalar3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_type = __scalar_as_int(my_postype.w);
    unsigned int my_body = d_body[my_pidx];
    Scalar my_diam = d_diameter[my_pidx];
    unsigned int my_head = d_head_list[my_pidx];

    Scalar3 f = box.makeFraction(my_pos, ghost_width);

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // number of available stencils
    unsigned int n_stencil = d_n_stencil[my_type];

    // index of current stencil (-1 to initialize)
    int cur_adj = -1;
    Scalar cell_dist2 = 0.0;

    // current cell (0 to initialize)
    unsigned int neigh_cell = 0;

    // size of current cell (0 to initialize)
    unsigned int neigh_size = 0;

    // current index in cell
    int cur_offset = threadIdx.x % threads_per_particle;

    bool done = false;

    // total number of neighbors
    unsigned int nneigh = 0;

    while (! done)
        {
        // initalize with default
        unsigned int neighbor;
        unsigned char has_neighbor = 0;

        // advance neighbor cell
        while (cur_offset >= neigh_size && !done )
            {
            cur_offset -= neigh_size;
            cur_adj++;

            if (cur_adj < n_stencil)
                {
                // compute the stenciled cell cartesian coordinates
                Scalar4 stencil = texFetchScalar4(d_stencil, stencil_1d_tex, stencil_idx(cur_adj, my_type));
                int sib = ib + __scalar_as_int(stencil.x);
                int sjb = jb + __scalar_as_int(stencil.y);
                int skb = kb + __scalar_as_int(stencil.z);
                cell_dist2 = stencil.w;

                // wrap through the boundary
                if (sib >= (int)ci.getW() && periodic.x) sib -= ci.getW();
                if (sib < 0 && periodic.x) sib += ci.getW();
                if (sjb >= (int)ci.getH() && periodic.y) sjb -= ci.getH();
                if (sjb < 0 && periodic.y) sjb += ci.getH();
                if (skb >= (int)ci.getD() && periodic.z) skb -= ci.getD();
                if (skb < 0 && periodic.z) skb += ci.getD();

                neigh_cell = ci(sib, sjb, skb);
                neigh_size = d_cell_size[neigh_cell];
                }
            else
                // we are past the end of the cell neighbors
                done = true;
            }

        // if the first thread in the cta has no work, terminate the loop
        if (done && !(threadIdx.x % threads_per_particle)) break;

        if (!done)
            {
            // use a do {} while(0) loop to process this particle so we can break for exclusions
            // in microbenchmarks, this is was faster than using bool exclude because it saved flops
            // it's a little easier to read than having 4 levels of if{} statements nested
            do
                {
                // read in the particle type (diameter and body as well while we've got the Scalar4 in)
                const Scalar4 neigh_tdb = texFetchScalar4(d_cell_tdb, cell_tdb_1d_tex, cli(cur_offset, neigh_cell));
                const unsigned int type_j = __scalar_as_int(neigh_tdb.x);
                const Scalar diam_j = neigh_tdb.y;
                const unsigned int body_j = __scalar_as_int(neigh_tdb.z);

                // skip any particles belonging to the same rigid body if requested
                if (filter_body && my_body != 0xffffffff && my_body == body_j) break;

                // compute the rlist based on the particle type we're interacting with
                Scalar r_list = s_r_list[typpair_idx(my_type,type_j)];
                if (r_list <= Scalar(0.0)) break;
                Scalar sqshift = Scalar(0.0);
                if (diameter_shift)
                    {
                    const Scalar delta = (my_diam + diam_j) * Scalar(0.5) - Scalar(1.0);
                    // r^2 < (r_list + delta)^2
                    // r^2 < r_listsq + delta^2 + 2*r_list*delta
                    sqshift = (delta + Scalar(2.0) * r_list) * delta;
                    }
                Scalar r_listsq = r_list*r_list + sqshift;

                // compare the check distance to the minimum cell distance, and pass without distance check if unnecessary
                if (cell_dist2 > r_listsq) break;

                // only load in the particle position and id if distance check is required
                const Scalar4 neigh_xyzf = texFetchScalar4(d_cell_xyzf, cell_xyzf_1d_tex, cli(cur_offset, neigh_cell));
                const Scalar3 neigh_pos = make_scalar3(neigh_xyzf.x, neigh_xyzf.y, neigh_xyzf.z);
                unsigned int cur_neigh = __scalar_as_int(neigh_xyzf.w);

                // a particle cannot neighbor itself
                if (my_pidx == (int)cur_neigh) break;

                Scalar3 dx = my_pos - neigh_pos;
                dx = box.minImage(dx);

                Scalar dr_sq = dot(dx,dx);

                if (dr_sq <= r_listsq)
                    {
                    neighbor = cur_neigh;
                    has_neighbor = 1;
                    }
                } while (0); // particle is processed exactly once

            // advance cur_offset
            cur_offset += threads_per_particle;
            }

        // no syncthreads here, we assume threads_per_particle < warp size

        // scan over flags
        int k = 0;
        #if (__CUDA_ARCH__ >= 300)
        unsigned char n = 1;
        k = warp_scan_sm30_stencil<threads_per_particle>::Scan(threadIdx.x % threads_per_particle, has_neighbor, &n);
        #endif

        if (has_neighbor && (nneigh + k) < s_Nmax[my_type])
            d_nlist[my_head + nneigh + k] = neighbor;

        // increment total neighbor count
        #if (__CUDA_ARCH__ >= 300)
        nneigh += n;
        #else
        if (has_neighbor)
            nneigh++;
        #endif
        } // end while

    if (threadIdx.x % threads_per_particle == 0)
        {
        // flag if we need to grow the neighbor list
        if (nneigh >= s_Nmax[my_type])
            atomicMax(&d_conditions[my_type], nneigh);

        d_n_neigh[my_pidx] = nneigh;
        d_last_updated_pos[my_pidx] = my_postype;
        }
    }

//! determine maximum possible block size
template<typename T>
int get_max_block_size_stencil(T func)
    {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))func);
    int max_threads = attr.maxThreadsPerBlock;
    // number of threads has to be multiple of warp size
    max_threads -= max_threads % max_threads_per_particle;
    return max_threads;
    }

//! Bind the textures on sm <= 30
/*!
 * \param d_cell_xyzf Cell list particle array
 * \param d_cell_tdb Cell list type-diameter-body array
 * \param n_elements Number of elements in the cell list arrays
 * \param d_stencil Stencil offset array
 * \param n_stencil_elements Number of elements in the stencil offset array
 */
void gpu_nlist_stencil_bind_texture(const Scalar4 *d_cell_xyzf,
                                    const Scalar4 *d_cell_tdb,
                                    unsigned int n_elements,
                                    const Scalar4 *d_stencil,
                                    unsigned int n_stencil_elements)
    {
    // bind the position texture
    cell_xyzf_1d_tex.normalized = false;
    cell_xyzf_1d_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, cell_xyzf_1d_tex, d_cell_xyzf, sizeof(Scalar4)*n_elements);

    // bind the position texture
    cell_tdb_1d_tex.normalized = false;
    cell_tdb_1d_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, cell_tdb_1d_tex, d_cell_tdb, sizeof(Scalar4)*n_elements);

    // bind the stencil texture
    stencil_1d_tex.normalized = false;
    stencil_1d_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, stencil_1d_tex, d_stencil, sizeof(Scalar4)*n_stencil_elements);
    }

//! recursive template to launch neighborlist with given template parameters
/* \tparam cur_tpp Number of threads per particle (assumed to be power of two) */
template<int cur_tpp>
inline void stencil_launcher(unsigned int *d_nlist,
                             unsigned int *d_n_neigh,
                             Scalar4 *d_last_updated_pos,
                             unsigned int *d_conditions,
                             const unsigned int *d_Nmax,
                             const unsigned int *d_head_list,
                             const unsigned int *d_pid_map,
                             const Scalar4 *d_pos,
                             const unsigned int *d_body,
                             const Scalar *d_diameter,
                             const unsigned int N,
                             const unsigned int *d_cell_size,
                             const Scalar4 *d_cell_xyzf,
                             const Scalar4 *d_cell_tdb,
                             const Index3D& ci,
                             const Index2D& cli,
                             const Scalar4 *d_stencil,
                             const unsigned int *d_n_stencil,
                             const Index2D& stencil_idx,
                             const BoxDim& box,
                             const Scalar *d_r_cut,
                             const Scalar r_buff,
                             const unsigned int ntypes,
                             const Scalar3& ghost_width,
                             bool filter_body,
                             bool diameter_shift,
                             const unsigned int threads_per_particle,
                             const unsigned int block_size,
                             const unsigned int compute_capability)
    {
    // shared memory = r_listsq + Nmax + stuff needed for neighborlist (computed below)
    Index2D typpair_idx(ntypes);
    unsigned int shared_size = sizeof(Scalar)*typpair_idx.getNumElements() + sizeof(unsigned int)*ntypes;

    if (threads_per_particle == cur_tpp && cur_tpp != 0)
        {
        if (!diameter_shift && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<0,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_stencil_bind_texture(d_cell_xyzf,
                                                                        d_cell_tdb,
                                                                        cli.getNumElements(),
                                                                        d_stencil,
                                                                        stencil_idx.getNumElements());

            unsigned int run_block_size = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size/threads_per_particle) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_stencil_kernel<0,cur_tpp><<<grid,run_block_size,shared_size>>>(d_nlist,
                                                                                             d_n_neigh,
                                                                                             d_last_updated_pos,
                                                                                             d_conditions,
                                                                                             d_Nmax,
                                                                                             d_head_list,
                                                                                             d_pid_map,
                                                                                             d_pos,
                                                                                             d_body,
                                                                                             d_diameter,
                                                                                             N,
                                                                                             d_cell_size,
                                                                                             d_cell_xyzf,
                                                                                             d_cell_tdb,
                                                                                             ci,
                                                                                             cli,
                                                                                             d_stencil,
                                                                                             d_n_stencil,
                                                                                             stencil_idx,
                                                                                             box,
                                                                                             d_r_cut,
                                                                                             r_buff,
                                                                                             ntypes,
                                                                                             ghost_width);
            }
        else if (!diameter_shift && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<1,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_stencil_bind_texture(d_cell_xyzf,
                                                                        d_cell_tdb,
                                                                        cli.getNumElements(),
                                                                        d_stencil,
                                                                        stencil_idx.getNumElements());

            unsigned int run_block_size = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size/threads_per_particle) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_stencil_kernel<1,cur_tpp><<<grid,run_block_size,shared_size>>>(d_nlist,
                                                                                             d_n_neigh,
                                                                                             d_last_updated_pos,
                                                                                             d_conditions,
                                                                                             d_Nmax,
                                                                                             d_head_list,
                                                                                             d_pid_map,
                                                                                             d_pos,
                                                                                             d_body,
                                                                                             d_diameter,
                                                                                             N,
                                                                                             d_cell_size,
                                                                                             d_cell_xyzf,
                                                                                             d_cell_tdb,
                                                                                             ci,
                                                                                             cli,
                                                                                             d_stencil,
                                                                                             d_n_stencil,
                                                                                             stencil_idx,
                                                                                             box,
                                                                                             d_r_cut,
                                                                                             r_buff,
                                                                                             ntypes,
                                                                                             ghost_width);
            }
        else if (diameter_shift && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<2,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_stencil_bind_texture(d_cell_xyzf,
                                                                        d_cell_tdb,
                                                                        cli.getNumElements(),
                                                                        d_stencil,
                                                                        stencil_idx.getNumElements());

            unsigned int run_block_size = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size/threads_per_particle) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_stencil_kernel<2,cur_tpp><<<grid,run_block_size,shared_size>>>(d_nlist,
                                                                                             d_n_neigh,
                                                                                             d_last_updated_pos,
                                                                                             d_conditions,
                                                                                             d_Nmax,
                                                                                             d_head_list,
                                                                                             d_pid_map,
                                                                                             d_pos,
                                                                                             d_body,
                                                                                             d_diameter,
                                                                                             N,
                                                                                             d_cell_size,
                                                                                             d_cell_xyzf,
                                                                                             d_cell_tdb,
                                                                                             ci,
                                                                                             cli,
                                                                                             d_stencil,
                                                                                             d_n_stencil,
                                                                                             stencil_idx,
                                                                                             box,
                                                                                             d_r_cut,
                                                                                             r_buff,
                                                                                             ntypes,
                                                                                             ghost_width);
            }
        else if (diameter_shift && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size_stencil(gpu_compute_nlist_stencil_kernel<3,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_stencil_bind_texture(d_cell_xyzf,
                                                                        d_cell_tdb,
                                                                        cli.getNumElements(),
                                                                        d_stencil,
                                                                        stencil_idx.getNumElements());

            unsigned int run_block_size = (block_size < max_block_size) ? block_size : max_block_size;
            dim3 grid(N / (block_size/threads_per_particle) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_stencil_kernel<3,cur_tpp><<<grid,run_block_size,shared_size>>>(d_nlist,
                                                                                             d_n_neigh,
                                                                                             d_last_updated_pos,
                                                                                             d_conditions,
                                                                                             d_Nmax,
                                                                                             d_head_list,
                                                                                             d_pid_map,
                                                                                             d_pos,
                                                                                             d_body,
                                                                                             d_diameter,
                                                                                             N,
                                                                                             d_cell_size,
                                                                                             d_cell_xyzf,
                                                                                             d_cell_tdb,
                                                                                             ci,
                                                                                             cli,
                                                                                             d_stencil,
                                                                                             d_n_stencil,
                                                                                             stencil_idx,
                                                                                             box,
                                                                                             d_r_cut,
                                                                                             r_buff,
                                                                                             ntypes,
                                                                                             ghost_width);
            }
        }
    else
        {
        stencil_launcher<cur_tpp/2>(d_nlist,
                                    d_n_neigh,
                                    d_last_updated_pos,
                                    d_conditions,
                                    d_Nmax,
                                    d_head_list,
                                    d_pid_map,
                                    d_pos,
                                    d_body,
                                    d_diameter,
                                    N,
                                    d_cell_size,
                                    d_cell_xyzf,
                                    d_cell_tdb,
                                    ci,
                                    cli,
                                    d_stencil,
                                    d_n_stencil,
                                    stencil_idx,
                                    box,
                                    d_r_cut,
                                    r_buff,
                                    ntypes,
                                    ghost_width,
                                    filter_body,
                                    diameter_shift,
                                    threads_per_particle,
                                    block_size,
                                    compute_capability);
        }
    }

//! template specialization to terminate recursion
template<>
inline void stencil_launcher<min_threads_per_particle/2>(unsigned int *d_nlist,
                                                         unsigned int *d_n_neigh,
                                                         Scalar4 *d_last_updated_pos,
                                                         unsigned int *d_conditions,
                                                         const unsigned int *d_Nmax,
                                                         const unsigned int *d_head_list,
                                                         const unsigned int *d_pid_map,
                                                         const Scalar4 *d_pos,
                                                         const unsigned int *d_body,
                                                         const Scalar *d_diameter,
                                                         const unsigned int N,
                                                         const unsigned int *d_cell_size,
                                                         const Scalar4 *d_cell_xyzf,
                                                         const Scalar4 *d_cell_tdb,
                                                         const Index3D& ci,
                                                         const Index2D& cli,
                                                         const Scalar4 *d_stencil,
                                                         const unsigned int *d_n_stencil,
                                                         const Index2D& stencil_idx,
                                                         const BoxDim& box,
                                                         const Scalar *d_r_cut,
                                                         const Scalar r_buff,
                                                         const unsigned int ntypes,
                                                         const Scalar3& ghost_width,
                                                         bool filter_body,
                                                         bool diameter_shift,
                                                         const unsigned int threads_per_particle,
                                                         const unsigned int block_size,
                                                         const unsigned int compute_capability)
    { }

hipError_t gpu_compute_nlist_stencil(unsigned int *d_nlist,
                                      unsigned int *d_n_neigh,
                                      Scalar4 *d_last_updated_pos,
                                      unsigned int *d_conditions,
                                      const unsigned int *d_Nmax,
                                      const unsigned int *d_head_list,
                                      const unsigned int *d_pid_map,
                                      const Scalar4 *d_pos,
                                      const unsigned int *d_body,
                                      const Scalar *d_diameter,
                                      const unsigned int N,
                                      const unsigned int *d_cell_size,
                                      const Scalar4 *d_cell_xyzf,
                                      const Scalar4 *d_cell_tdb,
                                      const Index3D& ci,
                                      const Index2D& cli,
                                      const Scalar4 *d_stencil,
                                      const unsigned int *d_n_stencil,
                                      const Index2D& stencil_idx,
                                      const BoxDim& box,
                                      const Scalar *d_r_cut,
                                      const Scalar r_buff,
                                      const unsigned int ntypes,
                                      const Scalar3& ghost_width,
                                      bool filter_body,
                                      bool diameter_shift,
                                      const unsigned int threads_per_particle,
                                      const unsigned int block_size,
                                      const unsigned int compute_capability)
    {
    stencil_launcher<max_threads_per_particle>(d_nlist,
                                               d_n_neigh,
                                               d_last_updated_pos,
                                               d_conditions,
                                               d_Nmax,
                                               d_head_list,
                                               d_pid_map,
                                               d_pos,
                                               d_body,
                                               d_diameter,
                                               N,
                                               d_cell_size,
                                               d_cell_xyzf,
                                               d_cell_tdb,
                                               ci,
                                               cli,
                                               d_stencil,
                                               d_n_stencil,
                                               stencil_idx,
                                               box,
                                               d_r_cut,
                                               r_buff,
                                               ntypes,
                                               ghost_width,
                                               filter_body,
                                               diameter_shift,
                                               threads_per_particle,
                                               block_size,
                                               compute_capability);
    return hipSuccess;
    }

/*!
 * \param d_pids Unsorted particle indexes
 * \param d_types Unsorted particle types
 * \param d_pos Particle position array
 * \param N Number of particles
 *
 * \a d_pids and \a d_types are trivially initialized to their current (unsorted) values. They are later sorted in
 * gpu_compute_nlist_stencil_sort_types().
 */
__global__ void gpu_compute_nlist_stencil_fill_types_kernel(unsigned int *d_pids,
                                                            unsigned int *d_types,
                                                            const Scalar4 *d_pos,
                                                            const unsigned int N)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    Scalar4 pos_i = d_pos[idx];
    unsigned int type = __scalar_as_int(pos_i.w);
    d_types[idx] = type;
    d_pids[idx] = idx;
    }

/*!
 * \param d_pids Unsorted particle indexes
 * \param d_types Unsorted particle types
 * \param d_pos Particle position array
 * \param N Number of particles
 */
hipError_t gpu_compute_nlist_stencil_fill_types(unsigned int *d_pids,
                                                 unsigned int *d_types,
                                                 const Scalar4 *d_pos,
                                                 const unsigned int N)
    {
    const unsigned int block_size = 128;

    gpu_compute_nlist_stencil_fill_types_kernel<<<N/block_size + 1, block_size>>>(d_pids, d_types, d_pos, N);

    return hipSuccess;
    }

/*!
 * \param d_pids Array of unsorted particle indexes
 * \param d_pids_alt Double buffer for particle indexes
 * \param d_types Array of unsorted particle types
 * \param d_types_alt Double buffer for particle types
 * \param d_tmp_storage Temporary allocation for sorting
 * \param tmp_storage_bytes Size of temporary allocation
 * \param swap Flag to swap the sorted particle indexes into the correct buffer
 * \param N number of particles
 *
 * This wrapper calls the CUB radix sorting methods, and so it needs to be called twice. Initially, \a d_tmp_storage
 * should be NULL, and the necessary temporary storage is saved into \a tmp_storage_bytes. This space must then be
 * allocated into \a d_tmp_storage, and on the second call, the sorting is performed.
 */
void gpu_compute_nlist_stencil_sort_types(unsigned int *d_pids,
                                          unsigned int *d_pids_alt,
                                          unsigned int *d_types,
                                          unsigned int *d_types_alt,
                                          void *d_tmp_storage,
                                          size_t &tmp_storage_bytes,
                                          bool &swap,
                                          const unsigned int N)
    {
    hipcub::DoubleBuffer<unsigned int> d_keys(d_types, d_types_alt);
    hipcub::DoubleBuffer<unsigned int> d_vals(d_pids, d_pids_alt);
    hipcub::DeviceRadixSort::SortPairs(d_tmp_storage, tmp_storage_bytes, d_keys, d_vals, N);
    if (d_tmp_storage != NULL)
        {
        swap = (d_vals.selector == 1);
        }
    }
