#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "NeighborListGPUBinned.cuh"
#include "hoomd/TextureTools.h"

/*! \file NeighborListGPUBinned.cu
    \brief Defines GPU kernel code for O(N) neighbor list generation on the GPU
*/

//! Texture for reading d_cell_xyzf
scalar4_tex_t cell_xyzf_1d_tex;

//! Warp-centric scan (Kepler and later)
template<int NT>
struct warp_scan_sm30
    {
    __device__ static int Scan(int tid, unsigned char x, unsigned char* total)
        {
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));

        int first = laneid - tid;

        #pragma unroll
        for(int offset = 1; offset < NT; offset += offset)
            {
            int y = __shfl(x,(first + tid - offset) &(WARP_SIZE -1));
            if(tid >= offset) x += y;
            }

        // all threads get the total from the last thread in the cta
        *total = __shfl(x,first + NT - 1);

        // shift by one (exclusive scan)
        int y = __shfl(x,(first + tid - 1) &(WARP_SIZE-1));
        x = tid ? y : 0;

        return x;
        }
    };

//! Kernel call for generating neighbor list on the GPU (Kepler optimized version)
/*! \tparam flags Set bit 1 to enable body filtering. Set bit 2 to enable diameter filtering.
    \param d_nlist Neighbor list data structure to write
    \param d_n_neigh Number of neighbors to write
    \param d_last_updated_pos Particle positions at this update are written to this array
    \param d_conditions Conditions array for writing overflow condition
    \param d_Nmax Maximum number of neighbors per type
    \param d_head_list List of indexes to access \a d_nlist
    \param d_pos Particle positions
    \param d_body Particle body indices
    \param d_diameter Particle diameters
    \param N Number of particles
    \param d_cell_size Number of particles in each cell
    \param d_cell_xyzf Cell contents (xyzf array from CellList with flag=type)
    \param d_cell_tdb Cell contents (tdb array from CellList with)
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer for indexing cells
    \param cli Cell list indexer for indexing into d_cell_xyzf
    \param cadji Adjacent cell indexer listing the 27 neighboring cells
    \param box Simulation box dimensions
    \param d_r_cut Cutoff radius stored by pair type r_cut(i,j)
    \param r_buff The maximum radius for which to include particles as neighbors
    \param ntypes Number of particle types
    \param ghost_width Width of ghost cell layer

    \note optimized for Kepler
*/
template<unsigned char flags, int threads_per_particle>
__global__ void gpu_compute_nlist_binned_kernel(unsigned int *d_nlist,
                                                    unsigned int *d_n_neigh,
                                                    Scalar4 *d_last_updated_pos,
                                                    unsigned int *d_conditions,
                                                    const unsigned int *d_Nmax,
                                                    const unsigned int *d_head_list,
                                                    const Scalar4 *d_pos,
                                                    const unsigned int *d_body,
                                                    const Scalar *d_diameter,
                                                    const unsigned int N,
                                                    const unsigned int *d_cell_size,
                                                    const Scalar4 *d_cell_xyzf,
                                                    const Scalar4 *d_cell_tdb,
                                                    const unsigned int *d_cell_adj,
                                                    const Index3D ci,
                                                    const Index2D cli,
                                                    const Index2D cadji,
                                                    const BoxDim box,
                                                    const Scalar *d_r_cut,
                                                    const Scalar r_buff,
                                                    const unsigned int ntypes,
                                                    const Scalar3 ghost_width)
    {
    bool filter_body = flags & 1;
    bool diameter_shift = flags & 2;

    // cache the r_listsq parameters into shared memory
    Index2D typpair_idx(ntypes);
    const unsigned int num_typ_parameters = typpair_idx.getNumElements();

    // shared data for per type pair parameters
    extern __shared__ unsigned char s_data[];

    // pointer for the r_listsq data
    Scalar *s_r_list = (Scalar *)(&s_data[0]);
    unsigned int *s_Nmax = (unsigned int *)(&s_data[sizeof(Scalar)*num_typ_parameters]);

    // load in the per type pair r_list
    for (unsigned int cur_offset = 0; cur_offset < num_typ_parameters; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < num_typ_parameters)
            {
            Scalar r_cut = d_r_cut[cur_offset + threadIdx.x];
            // force the r_list(i,j) to a skippable value if r_cut(i,j) is skippable
            s_r_list[cur_offset + threadIdx.x] = (r_cut > Scalar(0.0)) ? r_cut+r_buff : Scalar(-1.0);
            }
        if (cur_offset + threadIdx.x < ntypes)
            {
            s_Nmax[cur_offset + threadIdx.x] = d_Nmax[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();

    // each set of threads_per_particle threads is going to compute the neighbor list for a single particle
    int my_pidx;
    if (gridDim.y > 1)
        {
        // fermi workaround
        my_pidx = (blockIdx.x + blockIdx.y*65535) * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }
    else
        {
        my_pidx = blockIdx.x * (blockDim.x/threads_per_particle) + threadIdx.x/threads_per_particle;
        }

    // one thread per particle
    if (my_pidx >= N) return;

    Scalar4 my_postype = d_pos[my_pidx];
    Scalar3 my_pos = make_scalar3(my_postype.x, my_postype.y, my_postype.z);

    unsigned int my_type = __scalar_as_int(my_postype.w);
    unsigned int my_body = d_body[my_pidx];
    Scalar my_diam = d_diameter[my_pidx];
    unsigned int my_head = d_head_list[my_pidx];

    Scalar3 f = box.makeFraction(my_pos, ghost_width);

    // find the bin each particle belongs in
    int ib = (int)(f.x * ci.getW());
    int jb = (int)(f.y * ci.getH());
    int kb = (int)(f.z * ci.getD());

    uchar3 periodic = box.getPeriodic();

    // need to handle the case where the particle is exactly at the box hi
    if (ib == ci.getW() && periodic.x)
        ib = 0;
    if (jb == ci.getH() && periodic.y)
        jb = 0;
    if (kb == ci.getD() && periodic.z)
        kb = 0;

    int my_cell = ci(ib,jb,kb);

    // index of current neighbor
    unsigned int cur_adj = 0;

    // current cell
    unsigned int neigh_cell = d_cell_adj[cadji(cur_adj, my_cell)];

    // size of current cell
    unsigned int neigh_size = d_cell_size[neigh_cell];

    // current index in cell
    int cur_offset = threadIdx.x % threads_per_particle;

    bool done = false;

    // total number of neighbors
    unsigned int nneigh = 0;

    while (! done)
        {
        // initalize with default
        unsigned int neighbor;
        unsigned char has_neighbor = 0;

        // advance neighbor cell
        while (cur_offset >= neigh_size && !done )
            {
            cur_offset -= neigh_size;
            cur_adj++;
            if (cur_adj < cadji.getW())
                {
                neigh_cell = d_cell_adj[cadji(cur_adj, my_cell)];
                neigh_size = d_cell_size[neigh_cell];
                }
            else
                // we are past the end of the cell neighbors
                done = true;
            }

        // if the first thread in the cta has no work, terminate the loop
        if (done && !(threadIdx.x % threads_per_particle)) break;

        if (!done)
            {
            Scalar4 cur_xyzf = texFetchScalar4(d_cell_xyzf, cell_xyzf_1d_tex, cli(cur_offset, neigh_cell));

            Scalar4 cur_tdb = d_cell_tdb[cli(cur_offset, neigh_cell)];

            // advance cur_offset
            cur_offset += threads_per_particle;

            unsigned int neigh_type = __scalar_as_int(cur_tdb.x);

            // Only do the hard work if the particle should be included by r_cut(i,j)
            Scalar r_list = s_r_list[typpair_idx(my_type,neigh_type)];
            if (r_list > Scalar(0.0))
                {
                Scalar neigh_diam = cur_tdb.y;
                unsigned int neigh_body = __scalar_as_int(cur_tdb.z);

                Scalar3 neigh_pos = make_scalar3(cur_xyzf.x,
                                               cur_xyzf.y,
                                               cur_xyzf.z);
                int cur_neigh = __scalar_as_int(cur_xyzf.w);

                // compute the distance between the two particles
                Scalar3 dx = my_pos - neigh_pos;

                // wrap the periodic boundary conditions
                dx = box.minImage(dx);

                // compute dr squared
                Scalar drsq = dot(dx,dx);

                bool excluded = (my_pidx == cur_neigh);

                if (filter_body && my_body != 0xffffffff)
                    excluded = excluded | (my_body == neigh_body);

                Scalar sqshift = Scalar(0.0);
                if (diameter_shift)
                    {
                    const Scalar delta = (my_diam + neigh_diam) * Scalar(0.5) - Scalar(1.0);
                    // r^2 < (r_list + delta)^2
                    // r^2 < r_listsq + delta^2 + 2*r_list*delta
                    sqshift = (delta + Scalar(2.0) * r_list) * delta;
                    }

                // store result in shared memory
                if (drsq <= (r_list*r_list + sqshift) && !excluded)
                    {
                    neighbor = cur_neigh;
                    has_neighbor = 1;
                    }
                }
            }

        // no syncthreads here, we assume threads_per_particle < warp size

        // scan over flags
        int k = 0;
        #if (__CUDA_ARCH__ >= 300)
        unsigned char n = 1;
        k = warp_scan_sm30<threads_per_particle>::Scan(threadIdx.x % threads_per_particle, has_neighbor, &n);
        #endif

        if (has_neighbor && (nneigh + k) < s_Nmax[my_type])
            d_nlist[my_head + nneigh + k] = neighbor;

        // increment total neighbor count
        #if (__CUDA_ARCH__ >= 300)
        nneigh += n;
        #else
        if (has_neighbor)
            nneigh++;
        #endif
        } // end while

    if (threadIdx.x % threads_per_particle == 0)
        {
        // flag if we need to grow the neighbor list
        if (nneigh >= s_Nmax[my_type])
            atomicMax(&d_conditions[my_type], nneigh);

        d_n_neigh[my_pidx] = nneigh;
        d_last_updated_pos[my_pidx] = my_postype;
        }
    }

//! determine maximum possible block size
template<typename T>
int get_max_block_size(T func)
    {
    hipFuncAttributes attr;
    hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void*))func);
    int max_threads = attr.maxThreadsPerBlock;
    // number of threads has to be multiple of warp size
    max_threads -= max_threads % max_threads_per_particle;
    return max_threads;
    }

void gpu_nlist_binned_bind_texture(const Scalar4 *d_cell_xyzf, unsigned int n_elements)
    {
    // bind the position texture
    cell_xyzf_1d_tex.normalized = false;
    cell_xyzf_1d_tex.filterMode = hipFilterModePoint;
    hipBindTexture(0, cell_xyzf_1d_tex, d_cell_xyzf, sizeof(Scalar4)*n_elements);
    }

//! recursive template to launch neighborlist with given template parameters
/* \tparam cur_tpp Number of threads per particle (assumed to be power of two) */
template<int cur_tpp>
inline void launcher(unsigned int *d_nlist,
              unsigned int *d_n_neigh,
              Scalar4 *d_last_updated_pos,
              unsigned int *d_conditions,
              const unsigned int *d_Nmax,
              const unsigned int *d_head_list,
              const Scalar4 *d_pos,
              const unsigned int *d_body,
              const Scalar *d_diameter,
              const unsigned int N,
              const unsigned int *d_cell_size,
              const Scalar4 *d_cell_xyzf,
              const Scalar4 *d_cell_tdb,
              const unsigned int *d_cell_adj,
              const Index3D ci,
              const Index2D cli,
              const Index2D cadji,
              const BoxDim box,
              const Scalar *d_r_cut,
              const Scalar r_buff,
              const unsigned int ntypes,
              const Scalar3 ghost_width,
              const unsigned int compute_capability,
              unsigned int tpp,
              bool filter_body,
              bool diameter_shift,
              unsigned int block_size)
    {
    // shared memory = r_listsq + Nmax + stuff needed for neighborlist (computed below)
    Index2D typpair_idx(ntypes);
    unsigned int shared_size = sizeof(Scalar)*typpair_idx.getNumElements() + sizeof(unsigned int)*ntypes;

    if (tpp == cur_tpp && cur_tpp != 0)
        {
        if (!diameter_shift && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_kernel<0,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_binned_kernel<0,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                                         d_n_neigh,
                                                                                         d_last_updated_pos,
                                                                                         d_conditions,
                                                                                         d_Nmax,
                                                                                         d_head_list,
                                                                                         d_pos,
                                                                                         d_body,
                                                                                         d_diameter,
                                                                                         N,
                                                                                         d_cell_size,
                                                                                         d_cell_xyzf,
                                                                                         d_cell_tdb,
                                                                                         d_cell_adj,
                                                                                         ci,
                                                                                         cli,
                                                                                         cadji,
                                                                                         box,
                                                                                         d_r_cut,
                                                                                         r_buff,
                                                                                         ntypes,
                                                                                         ghost_width);
            }
        else if (!diameter_shift && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_kernel<1,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }

            gpu_compute_nlist_binned_kernel<1,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                                         d_n_neigh,
                                                                                         d_last_updated_pos,
                                                                                         d_conditions,
                                                                                         d_Nmax,
                                                                                         d_head_list,
                                                                                         d_pos,
                                                                                         d_body,
                                                                                         d_diameter,
                                                                                         N,
                                                                                         d_cell_size,
                                                                                         d_cell_xyzf,
                                                                                         d_cell_tdb,
                                                                                         d_cell_adj,
                                                                                         ci,
                                                                                         cli,
                                                                                         cadji,
                                                                                         box,
                                                                                         d_r_cut,
                                                                                         r_buff,
                                                                                         ntypes,
                                                                                         ghost_width);
            }
        else if (diameter_shift && !filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_kernel<2,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }
            gpu_compute_nlist_binned_kernel<2,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                                         d_n_neigh,
                                                                                         d_last_updated_pos,
                                                                                         d_conditions,
                                                                                         d_Nmax,
                                                                                         d_head_list,
                                                                                         d_pos,
                                                                                         d_body,
                                                                                         d_diameter,
                                                                                         N,
                                                                                         d_cell_size,
                                                                                         d_cell_xyzf,
                                                                                         d_cell_tdb,
                                                                                         d_cell_adj,
                                                                                         ci,
                                                                                         cli,
                                                                                         cadji,
                                                                                         box,
                                                                                         d_r_cut,
                                                                                         r_buff,
                                                                                         ntypes,
                                                                                         ghost_width);
            }
        else if (diameter_shift && filter_body)
            {
            static unsigned int max_block_size = UINT_MAX;
            if (max_block_size == UINT_MAX)
                max_block_size = get_max_block_size(gpu_compute_nlist_binned_kernel<3,cur_tpp>);
            if (compute_capability < 35) gpu_nlist_binned_bind_texture(d_cell_xyzf, cli.getNumElements());

            block_size = block_size < max_block_size ? block_size : max_block_size;
            dim3 grid(N / (block_size/tpp) + 1);
            if (compute_capability < 30 && grid.x > 65535)
                {
                grid.y = grid.x/65535 + 1;
                grid.x = 65535;
                }
            gpu_compute_nlist_binned_kernel<3,cur_tpp><<<grid, block_size,shared_size>>>(d_nlist,
                                                                                         d_n_neigh,
                                                                                         d_last_updated_pos,
                                                                                         d_conditions,
                                                                                         d_Nmax,
                                                                                         d_head_list,
                                                                                         d_pos,
                                                                                         d_body,
                                                                                         d_diameter,
                                                                                         N,
                                                                                         d_cell_size,
                                                                                         d_cell_xyzf,
                                                                                         d_cell_tdb,
                                                                                         d_cell_adj,
                                                                                         ci,
                                                                                         cli,
                                                                                         cadji,
                                                                                         box,
                                                                                         d_r_cut,
                                                                                         r_buff,
                                                                                         ntypes,
                                                                                         ghost_width);
            }
        }
    else
        {
        launcher<cur_tpp/2>(d_nlist,
                     d_n_neigh,
                     d_last_updated_pos,
                     d_conditions,
                     d_Nmax,
                     d_head_list,
                     d_pos,
                     d_body,
                     d_diameter,
                     N,
                     d_cell_size,
                     d_cell_xyzf,
                     d_cell_tdb,
                     d_cell_adj,
                     ci,
                     cli,
                     cadji,
                     box,
                     d_r_cut,
                     r_buff,
                     ntypes,
                     ghost_width,
                     compute_capability,
                     tpp,
                     filter_body,
                     diameter_shift,
                     block_size
                     );
        }
    }

//! template specialization to terminate recursion
template<>
inline void launcher<min_threads_per_particle/2>(unsigned int *d_nlist,
              unsigned int *d_n_neigh,
              Scalar4 *d_last_updated_pos,
              unsigned int *d_conditions,
              const unsigned int *d_Nmax,
              const unsigned int *d_head_list,
              const Scalar4 *d_pos,
              const unsigned int *d_body,
              const Scalar *d_diameter,
              const unsigned int N,
              const unsigned int *d_cell_size,
              const Scalar4 *d_cell_xyzf,
              const Scalar4 *d_cell_tdb,
              const unsigned int *d_cell_adj,
              const Index3D ci,
              const Index2D cli,
              const Index2D cadji,
              const BoxDim box,
              const Scalar *d_r_cut,
              const Scalar r_buff,
              const unsigned int ntypes,
              const Scalar3 ghost_width,
              const unsigned int compute_capability,
              unsigned int tpp,
              bool filter_body,
              bool diameter_shift,
              unsigned int block_size)
    { }

hipError_t gpu_compute_nlist_binned(unsigned int *d_nlist,
                                     unsigned int *d_n_neigh,
                                     Scalar4 *d_last_updated_pos,
                                     unsigned int *d_conditions,
                                     const unsigned int *d_Nmax,
                                     const unsigned int *d_head_list,
                                     const Scalar4 *d_pos,
                                     const unsigned int *d_body,
                                     const Scalar *d_diameter,
                                     const unsigned int N,
                                     const unsigned int *d_cell_size,
                                     const Scalar4 *d_cell_xyzf,
                                     const Scalar4 *d_cell_tdb,
                                     const unsigned int *d_cell_adj,
                                     const Index3D& ci,
                                     const Index2D& cli,
                                     const Index2D& cadji,
                                     const BoxDim& box,
                                     const Scalar *d_r_cut,
                                     const Scalar r_buff,
                                     const unsigned int ntypes,
                                     const unsigned int threads_per_particle,
                                     const unsigned int block_size,
                                     bool filter_body,
                                     bool diameter_shift,
                                     const Scalar3& ghost_width,
                                     const unsigned int compute_capability)
    {
    launcher<max_threads_per_particle>(d_nlist,
                                   d_n_neigh,
                                   d_last_updated_pos,
                                   d_conditions,
                                   d_Nmax,
                                   d_head_list,
                                   d_pos,
                                   d_body,
                                   d_diameter,
                                   N,
                                   d_cell_size,
                                   d_cell_xyzf,
                                   d_cell_tdb,
                                   d_cell_adj,
                                   ci,
                                   cli,
                                   cadji,
                                   box,
                                   d_r_cut,
                                   r_buff,
                                   ntypes,
                                   ghost_width,
                                   compute_capability,
                                   threads_per_particle,
                                   filter_body,
                                   diameter_shift,
                                   block_size
                                   );

    return hipSuccess;
    }
