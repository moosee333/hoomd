#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: mphoward
#include "NeighborListGPUTree.cuh"
#include "hoomd/TextureTools.h"

#include "hoomd/extern/cub/hipcub/hipcub.hpp"

#define MORTON_CODE_BITS   30       //!< Length of the Morton code in bits (k = 10 bits per direction)
#define MORTON_CODE_N_BINS 1024     //!< Number of bins (2^10) per direction to generate 30 bit Morton codes
#define MORTON_TYPE_MASK_64 0x000000003fffffffu //!< 64 bit mask to turn morton code-type back to morton code

/*! \file NeighborListGPUTree.cu
    \brief Defines GPU kernel code for neighbor list tree traversal on the GPU
*/

//! Texture for reading particle positions
scalar4_tex_t pdata_pos_tex;
//! Texture for reading leaf data
scalar4_tex_t leaf_xyzf_tex;
//! Texture for the diameter / body
scalar2_tex_t leaf_db_tex;
//! Texture for reading node upper and lower bounds
scalar4_tex_t aabb_node_bounds_tex;
//! Texture for the head list
texture<unsigned int, 1, hipReadModeElementType> head_list_tex;


//!< Expands a 10-bit integer into 30 bits by inserting 2 zeros after each bit.
/*!
 * \param v unsigned integer with 10 bits set
 * \returns The integer expanded with two zeros interleaved between bits
 * http://devblogs.nvidia.com/parallelforall/thinking-parallel-part-iii-tree-construction-gpu/
 */
__device__ inline unsigned int expandBits(unsigned int v)
{
    v = (v * 0x00010001u) & 0xFF0000FFu;
    v = (v * 0x00000101u) & 0x0F00F00Fu;
    v = (v * 0x00000011u) & 0xC30C30C3u;
    v = (v * 0x00000005u) & 0x49249249u;
    return v;
}

//! Assigns the Morton code-type key for each particle on this processor
/*!
 * \param d_morton_types Morton code-type keys per particle
 * \param d_map_tree_pid List to be overwritten with particle ids in ascending order
 * \param d_morton_conditions Flag if a local particle (not a ghost) is detected out of bounds
 * \param d_pos Particle positions
 * \param N Number of local particles
 * \param nghosts Number of ghost particles
 * \param box Local simulation box
 * \param ghost_width Anticipated size of the ghost layer for nonbonded interactions
 *
 * \b Implementation
 * A sorting key is generated for each particle by determining the 30 bit Morton code for each particle, and then
 * concatenating onto the type. Both the Morton code and the type are 32 bit integers, so the concatenation is stored
 * compactly in a 64 bit integer morton_type = (type << 30) + morton code. In this way, a lexicographic sort will
 * sort first by type, then by morton code. The corresponding particle id (thread index) is stashed into d_map_tree_pid
 * to track particles after sorting.
 */
__global__ void gpu_nlist_morton_types_kernel(uint64_t *d_morton_types,
                                              unsigned int *d_map_tree_pid,
                                              int *d_morton_conditions,
                                              const Scalar4 *d_pos,
                                              const unsigned int N,
                                              const unsigned int nghosts,
                                              const BoxDim box,
                                              const Scalar3 ghost_width)
    {
    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= N+nghosts)
        return;

    // acquire particle data
    Scalar4 postype = d_pos[idx];
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    const unsigned int type = __scalar_as_int(postype.w);

    // get position in simulation box
    uchar3 periodic = box.getPeriodic();
    Scalar3 f = box.makeFraction(pos,ghost_width);

    /* check if the particle is inside the unit cell + ghost layer in all dimensions
     * this tolerance is small enough that when we multiply by the morton code bin size, we are still in range
     * we silently ignore ghosts outside of this width, and instead deal with that special case below
     * where extra ghosts are communicated (e.g. for bonded interactions)
     */
    if (((f.x < Scalar(-0.00001) || f.x >= Scalar(1.00001)) ||
        (f.y < Scalar(-0.00001) || f.y >= Scalar(1.00001)) ||
        (f.z < Scalar(-0.00001) || f.z >= Scalar(1.00001))) && idx < N)
        {
        atomicMax(d_morton_conditions,idx+1);
        return;
        }

    // find the bin each particle belongs in
    int ib = (int)(f.x * MORTON_CODE_N_BINS);
    int jb = (int)(f.y * MORTON_CODE_N_BINS);
    int kb = (int)(f.z * MORTON_CODE_N_BINS);

    if (!periodic.x) // ghosts exist and may be past layer width
        {
        // handle special cases where random ghosts are beyond the expected layer
        // by just rounding to the nearest edge
        if (ib < 0)
            {
            ib = 0;
            }
        else if (ib >= MORTON_CODE_N_BINS)
            {
            ib = MORTON_CODE_N_BINS - 1;
            }
        }
    else if (ib == MORTON_CODE_N_BINS) // some particles lie exactly on the edge, floor them to zero
        {
        ib = 0;
        }

    // do as for x in y
    if (!periodic.y)
        {
        if (jb < 0)
            {
            jb = 0;
            }
        else if (jb >= MORTON_CODE_N_BINS)
            {
            jb = MORTON_CODE_N_BINS - 1;
            }
        }
    else if (jb == MORTON_CODE_N_BINS)
        {
        jb = 0;
        }

    // do as for y in z
    if (!periodic.z)
        {
        if (kb < 0)
            {
            kb = 0;
            }
        else if (kb >= MORTON_CODE_N_BINS)
            {
            kb = MORTON_CODE_N_BINS - 1;
            }
        }
    else if (kb == MORTON_CODE_N_BINS)
        {
        kb = 0;
        }

    // inline call to some bit swizzling arithmetic
    unsigned int ii = expandBits((unsigned int)ib);
    unsigned int jj = expandBits((unsigned int)jb);
    unsigned int kk = expandBits((unsigned int)kb);
    unsigned int morton_code = ii * 4 + jj * 2 + kk;

    // save the morton code and corresponding particle index for sorting
    // the morton codes hold both the type and the code to sort by both type and position simultaneously
    d_morton_types[idx] = (((uint64_t)type) << MORTON_CODE_BITS) + (uint64_t)morton_code;
    d_map_tree_pid[idx] = idx;
    }

/*!
 * \param d_morton_types Morton code-type keys per particle
 * \param d_map_tree_pid List to be overwritten with particle ids in ascending order
 * \param d_morton_conditions Flag if a local particle (not a ghost) is detected out of bounds
 * \param d_pos Particle positions
 * \param N Number of local particles
 * \param nghosts Number of ghost particles
 * \param box Local simulation box
 * \param ghost_width Anticipated size of the ghost layer for nonbonded interactions
 * \param block_size Requested thread block size of kernel launch
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_morton_types(uint64_t *d_morton_types,
                                   unsigned int *d_map_tree_pid,
                                   int *d_morton_conditions,
                                   const Scalar4 *d_pos,
                                   const unsigned int N,
                                   const unsigned int nghosts,
                                   const BoxDim& box,
                                   const Scalar3 ghost_width,
                                   const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_morton_types_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);

    gpu_nlist_morton_types_kernel<<<(N+nghosts)/run_block_size + 1, run_block_size>>>(d_morton_types,
                                                                                      d_map_tree_pid,
                                                                                      d_morton_conditions,
                                                                                      d_pos,
                                                                                      N,
                                                                                      nghosts,
                                                                                      box,
                                                                                      ghost_width);
    return hipSuccess;
    }

/*!
 * \param d_morton_types Morton code-type keys per particle
 * \param d_morton_types_alt Auxiliary array of equal size to d_morton_types for double buffered sorting
 * \param d_map_tree_pid List of particle ids
 * \param d_map_tree_pid_alt Auxiliary array of equal size to d_map_tree_pid for double buffered sorting
 * \param d_tmp_storage Temporary storage in device memory
 * \param tmp_storage_bytes Number of bytes allocated for temporary storage
 * \param swap_morton Flag to switch real data from auxiliary array to primary array after sorting
 * \param swap_map Flag to switch real data from auxiliary array to primary array after sorting
 * \param Ntot Total number of keys to sort
 * \param n_type_bits Number of bits to check for particle types
 *
 * \returns hipSuccess on completion
 *
 * \b Implementation
 * The CUB library is used for device-wide radix sorting. Radix sorting is O(kN) where k is the number of bits to check
 * in an unsigned integer key, and N is the number of keys. We restrict the number of bits checked in the max 64 bit
 * keys by only checking up to the MORTON_CODE_BITS + n_type_bits most significant bit. CUB DeviceRadixSort performs
 * its own tuning at run time.
 *
 * Because CUB requires temporary storage, this function must be called twice. First, when \a d_tmp_storage is NULL,
 * the number of bytes required for temporary storage is saved in \a tmp_storage_bytes. This memory must then be
 * allocated in \a d_tmp_storage. On the second call, the radix sort is performed. Because the radix sort may put the
 * active (sorted) buffer in either slot of the DoubleBuffer, a boolean flag is set in \a swap_morton and \a swap_map
 * for whether these data arrays should be swapped.
 */
hipError_t gpu_nlist_morton_sort(uint64_t *d_morton_types,
                                  uint64_t *d_morton_types_alt,
                                  unsigned int *d_map_tree_pid,
                                  unsigned int *d_map_tree_pid_alt,
                                  void *d_tmp_storage,
                                  size_t &tmp_storage_bytes,
                                  bool &swap_morton,
                                  bool &swap_map,
                                  const unsigned int Ntot,
                                  const unsigned int n_type_bits)
    {
    // initialize memory as "double buffered"
    hipcub::DoubleBuffer<uint64_t> d_keys(d_morton_types, d_morton_types_alt);
    hipcub::DoubleBuffer<unsigned int> d_vals(d_map_tree_pid, d_map_tree_pid_alt);

    // on the first pass, this just sizes the temporary storage
    // on the second pass, it actually does the radix sort
    hipcub::DeviceRadixSort::SortPairs(d_tmp_storage,
                                    tmp_storage_bytes,
                                    d_keys,
                                    d_vals,
                                    Ntot,
                                    0,
                                    MORTON_CODE_BITS+n_type_bits);

    // we've only done something to the buffers on the second time when temporary storage is allocated
    if (d_tmp_storage != NULL)
        {
        // mark that the gpu arrays should be flipped if the final result is not in the right array
        swap_morton = (d_keys.selector == 1);
        swap_map = (d_vals.selector == 1);
        }

    return hipSuccess;
    }

//! Kernel to merge adjacent codes into leaf nodes
/*!
 * \param d_tree_aabbs Flat array holding all AABBs for the tree
 * \param d_morton_codes_red The Morton codes corresponding to the merged leafs
 * \param d_tree_parent_sib Parent and sibling indexes for all nodes
 * \param d_morton_types Morton-code type keys for all particles
 * \param d_pos Particle positions
 * \param d_num_per_type Number of particles per type
 * \param ntypes Number of particle types
 * \param d_map_tree_pid Sorted particle order (maps local index to ParticleData index)
 * \param d_leaf_offset Amount to subtract from the expected leaf starting index to make an array with no holes by type
 * \param d_type_head Index to first type and leaf ordered particles by type
 * \param Ntot Total number of keys to sort
 * \param nleafs Number of leaf nodes
 *
 * \b Implementation
 * One thread per leaf is called, and is responsible for merging NLIST_GPU_PARTICLES_PER_LEAF into an AABB. Each thread
 * first determines what type of leaf particle it is operating on by calculating and iterating on the number of leafs
 * of each type. Then, the starting index is determined by subtracting d_leaf_offset[type] from the starting index that
 * would be set in a nleaf x NLIST_GPU_PARTICLES_PER_LEAF array. The reason for this complexity is that the leaf particle
 * array is not permitted to have any "holes" in it for faster traversal. The AABB is merged from the particle
 * positions, and a Morton code is assigned to this AABB for determining tree hierarchy based on the Morton code of
 * the first particle in the leaf. Although this does not necessarily generate the best ordering along the Z order curve
 * for the newly merged leafs, it does guarantee that the leaf Morton codes are still in lexicographic ordering.
 *
 * AABBs are stored as two Scalar4s in a flat array. The first three coordinates of each Scalar4 correspond to the upper
 * and lower bounds of the AABB. The last value of the upper AABB will hold a "rope" for traversing the tree (see
 * gpu_nlist_bubble_aabbs_kernel), while the last value of the lower AABB holds the number of particles for a leaf node,
 * or the left child for an internal node. This is determined by setting a bit to mark this value as a rope or as child.
 */
__global__ void gpu_nlist_merge_particles_kernel(Scalar4 *d_tree_aabbs,
                                                 uint32_t *d_morton_codes_red,
                                                 uint2 *d_tree_parent_sib,
                                                 const uint64_t *d_morton_types,
                                                 const Scalar4 *d_pos,
                                                 const unsigned int *d_num_per_type,
                                                 const unsigned int ntypes,
                                                 const unsigned int *d_map_tree_pid,
                                                 const unsigned int *d_leaf_offset,
                                                 const unsigned int *d_type_head,
                                                 const unsigned int Ntot,
                                                 const unsigned int nleafs)
    {
    // leaf index
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per leaf
    if (idx >= nleafs)
        return;

    // get what type of leaf I am
    unsigned int total_bins = 0;
    int leaf_type = -1;
    unsigned int max_idx = Ntot;
    for (unsigned int cur_type=0; leaf_type == -1 && cur_type < ntypes; ++cur_type)
        {
        total_bins += (d_num_per_type[cur_type] + NLIST_GPU_PARTICLES_PER_LEAF - 1)/NLIST_GPU_PARTICLES_PER_LEAF;

        if (idx < total_bins)
            {
            leaf_type = cur_type;
            for (unsigned int next_type=cur_type+1; next_type < ntypes; ++next_type)
                {
                if (d_type_head[next_type])
                    {
                    max_idx = d_type_head[next_type] - 1;
                    break; // quit out of this inner loop once a match is found
                    }
                }
            break; // quit the outer loop
            }
        }

    // get the starting particle index assuming naive leaf structure, and then subtract offset to eliminate "holes"
    unsigned int start_idx = idx*NLIST_GPU_PARTICLES_PER_LEAF - d_leaf_offset[leaf_type];
    unsigned int end_idx = (max_idx - start_idx > NLIST_GPU_PARTICLES_PER_LEAF) ? start_idx + NLIST_GPU_PARTICLES_PER_LEAF : max_idx;


    // upper also holds the skip value, but we have no idea what this is right now
    Scalar4 upper = d_pos[ d_map_tree_pid[start_idx] ];
    upper.w = 0.0f;

    // lower holds the particle number, we have one already
    Scalar4 lower = upper;
    unsigned int npart = 1;

    for (unsigned int cur_p=start_idx+1; cur_p < end_idx; ++cur_p)
        {
        Scalar4 cur_pos = d_pos[ d_map_tree_pid[cur_p] ];

        // merge the boxes together
        if (cur_pos.x < lower.x) lower.x = cur_pos.x;
        if (cur_pos.x > upper.x) upper.x = cur_pos.x;

        if (cur_pos.y < lower.y) lower.y = cur_pos.y;
        if (cur_pos.y > upper.y) upper.y = cur_pos.y;

        if (cur_pos.z < lower.z) lower.z = cur_pos.z;
        if (cur_pos.z > upper.z) upper.z = cur_pos.z;

        ++npart;
        }

    d_tree_aabbs[2*idx] = upper;
    d_tree_aabbs[2*idx + 1] = make_scalar4(lower.x, lower.y, lower.z, __int_as_scalar(npart << 1));

    // take logical AND with the 30 bit mask for the morton codes to extract just the morton code
    // no sense swinging around 64 bit integers anymore
    d_morton_codes_red[idx] = (unsigned int)(d_morton_types[start_idx] & MORTON_TYPE_MASK_64);

    // fill the parent/sib relationships as if everything is a single leaf at first, to be overridden by hierarchy gen
    // when this is not the case
    d_tree_parent_sib[idx] = make_uint2(idx, idx << 1);
    }

/*!
 * \param d_tree_aabbs Flat array holding all AABBs for the tree
 * \param d_morton_codes_red The Morton codes corresponding to the merged leafs
 * \param d_tree_parent_sib Parent and sibling indexes for all nodes
 * \param d_morton_types Morton-code type keys for all particles
 * \param d_pos Particle positions
 * \param d_num_per_type Number of particles per type
 * \param ntypes Number of particle types
 * \param d_map_tree_pid Sorted particle order (maps local index to ParticleData index)
 * \param d_leaf_offset Amount to subtract from the expected leaf starting index to make an array with no holes by type
 * \param d_type_head Index to first type and leaf ordered particles by type
 * \param Ntot Total number of keys to sort
 * \param nleafs Number of leaf nodes
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_merge_particles(Scalar4 *d_tree_aabbs,
                                      uint32_t *d_morton_codes_red,
                                      uint2 *d_tree_parent_sib,
                                      const uint64_t *d_morton_types,
                                      const Scalar4 *d_pos,
                                      const unsigned int *d_num_per_type,
                                      const unsigned int ntypes,
                                      const unsigned int *d_map_tree_pid,
                                      const unsigned int *d_leaf_offset,
                                      const unsigned int *d_type_head,
                                      const unsigned int Ntot,
                                      const unsigned int nleafs,
                                      const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_merge_particles_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);

    gpu_nlist_merge_particles_kernel<<<nleafs/run_block_size + 1, block_size>>>(d_tree_aabbs,
                                                                                d_morton_codes_red,
                                                                                d_tree_parent_sib,
                                                                                d_morton_types,
                                                                                d_pos,
                                                                                d_num_per_type,
                                                                                ntypes,
                                                                                d_map_tree_pid,
                                                                                d_leaf_offset,
                                                                                d_type_head,
                                                                                Ntot,
                                                                                nleafs);
    return hipSuccess;
    }

//! Computes the longest common prefix between Morton codes
/*!
 * \param d_morton_codes Array of Morton codes
 * \param i First Morton code index
 * \param j Second Morton code index
 * \param min_idx The smallest index considered "in range" (inclusive)
 * \param max_idx The last index considered "in range" (inclusive)
 *
 * \returns number of bits shared between the Morton codes of i and j
 *
 * delta(i,j) is defined as the largest number of bits shared between Morton codes i and j. When the Morton codes are
 * sorted, this implies delta(i',j') >= delta(i,j) for any i',j' in [i,j]. If i and j lie outside
 * of the range of Morton codes corresponding to this tree, then it always returns -1. If the Morton codes for i and j
 * are identical, then the longest prefix of i and j is used as a tie breaker.
 */
__device__ inline int delta(const uint32_t *d_morton_codes,
                            unsigned int i,
                            unsigned int j,
                            int min_idx,
                            int max_idx)
    {
    if (j > max_idx || j < min_idx)
        {
        return -1;
        }

    uint32_t first_code = d_morton_codes[i];
    uint32_t last_code = d_morton_codes[j];

    // if codes match, then use index as tie breaker
    // the number of shared bits is equal to the 32 bits in the integer, plus the number of bits shared between the
    // indexes (offset from the start of the node range to make things simpler)
    if (first_code == last_code)
        {
        return (32 + __clz((i-min_idx) ^ (j-min_idx)));
        }
    else
        {
        return __clz(first_code ^ last_code);
        }
    }

//! Determines the range of Morton codes that a node covers
/*!
 * \param d_morton_codes Array of Morton codes
 * \param min_idx The smallest Morton code index considered "in range" (inclusive)
 * \param max_idx The last Morton code index considered "in range" (inclusive)
 * \param idx Current node (Morton code) index
 *
 * \returns the minimum and maximum leafs covered by this node
 * \note This is a literal implementation of the Karras pseudocode, with no optimizations or refinement.
 *       Tero Karras, "Maximizing parallelism in the construction of BVHs, octrees, and k-d trees",
 *       High Performance Graphics (2012).
 */
__device__ inline uint2 determineRange(const uint32_t *d_morton_codes,
                                       const int min_idx,
                                       const int max_idx,
                                       const int idx)
    {
    int forward_prefix = delta(d_morton_codes, idx, idx+1, min_idx, max_idx);
    int backward_prefix = delta(d_morton_codes, idx, idx-1, min_idx, max_idx);

    // get direction of the range based on sign
    int d = ((forward_prefix - backward_prefix) > 0) ? 1 : -1;

    // get minimum prefix
    int min_prefix = delta(d_morton_codes, idx, idx-d, min_idx, max_idx);

    // get maximum prefix by binary search
    int lmax = 2;
    while( delta(d_morton_codes, idx, idx + d*lmax, min_idx, max_idx) > min_prefix)
        {
        lmax = lmax << 1;
        }

    unsigned int len = 0;
    unsigned int step = lmax;
    do
        {
        step = step >> 1;
        unsigned int new_len = len + step;
        if (delta(d_morton_codes, idx, idx + d*new_len, min_idx, max_idx) > min_prefix)
            len = new_len;
        }
    while (step > 1);

   // order range based on direction
    uint2 range;
    if (d > 0)
        {
        range.x = idx;
        range.y = idx + len;
        }
    else
        {
        range.x = idx - len;
        range.y = idx;
        }
    return range;
    }

//! Finds the split position in Morton codes covered by a range
/*!
 * \param d_morton_codes Array of Morton codes
 * \param first First leaf node in the range
 * \param last Last leaf node in the range
 *
 * \returns the leaf index corresponding to the split in Morton codes
 * See determineRange for original source of algorithm.
 */
__device__ inline unsigned int findSplit(const uint32_t *d_morton_codes,
                                         const unsigned int first,
                                         const unsigned int last)
    {
    uint32_t first_code = d_morton_codes[first];
    uint32_t last_code = d_morton_codes[last];

    // if codes match, then just split evenly
    if (first_code == last_code)
        return (first + last) >> 1;

    // get the length of the common prefix
    int common_prefix = __clz(first_code ^ last_code);

    // assume split starts at first, and begin binary search
    unsigned int split = first;
    unsigned int step = last - first;
    do
        {
        // exponential decrease (is factor of 2 best?)
        step = (step + 1) >> 1;
        unsigned int new_split = split + step;

        // if proposed split lies within range
        if (new_split < last)
            {
            unsigned int split_code = d_morton_codes[new_split];
            int split_prefix = __clz(first_code ^ split_code);

            // if new split shares a longer number of bits, accept it
            if (split_prefix > common_prefix)
                {
                split = new_split;
                }
            }
        }
    while (step > 1);

    return split;
    }

//! Kernel to generate the parent-child-sibling relationships between nodes
/*!
 * \param d_tree_parent_sib Parent and sibling for each node in the tree
 * \param d_morton_codes Morton codes for each leaf node
 * \param d_num_per_type Number of particles per type
 * \param ntypes Number of types
 * \param nleafs Number of leafs
 *
 * \b Implementation
 * One thread is called per internal node in a single kernel launch. Each thread first determines its "local" index
 * as an internal node within a tree based on the number of leafs per tree. The range of leafs covered by the internal
 * node is determined, and then its split position is identified. The split identifies the children of the node as
 * another internal node or as a leaf node.
 *
 * The parent and sibling of each child node is saved. The sibling id is bit shifted so as to use a single bit to encode
 * the sibling as a right child or left child (after shifting, we set the bit to 1 if the sibling is a right child).
 * If the child is a root node, it also saves information for itself (since no other node ever identifies a root as a
 * child node).
 */
__global__ void gpu_nlist_gen_hierarchy_kernel(uint2 *d_tree_parent_sib,
                                               const uint32_t *d_morton_codes,
                                               const unsigned int *d_num_per_type,
                                               const unsigned int ntypes,
                                               const unsigned int nleafs,
                                               const unsigned int ninternal)
    {
    // compute the internal node index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per internal node
    if (idx >= ninternal)
        return;

    // get what type of leaf I am
    unsigned int min_idx = 0; // the "0" of the leaf node array
    unsigned int max_idx = 0; // the "N-1" of the leaf node array

    unsigned int node_idx = idx;
    unsigned int origin = 0;
    unsigned int end = 0;

    unsigned int cur_type=0;
    unsigned int active_types=0;
    for (cur_type=0; cur_type < ntypes; ++cur_type)
        {
        // current min index is the previous max index
        min_idx = max_idx;
        // max index adds the number of internal nodes in this type (nleaf - 1)
        const unsigned int cur_nleaf = (d_num_per_type[cur_type] + NLIST_GPU_PARTICLES_PER_LEAF - 1)/NLIST_GPU_PARTICLES_PER_LEAF;
        if (cur_nleaf > 0)
            {
            max_idx += cur_nleaf-1;
            ++active_types;
            }

        // we break the loop if we are in range
        if (idx < max_idx)
            {
            // decrement by 1 to get this back into the number we really need
            --active_types;

            // now, we repurpose the min and max index to now correspond to the *leaf* index.
            // the min index is the minimum *leaf* index
            origin = min_idx + active_types;
            end = max_idx + active_types;
            node_idx += active_types;
            break;
            }
        }

    // enact the magical split determining
    uint2 range = determineRange(d_morton_codes, origin, end, node_idx);
    unsigned int first = range.x;
    unsigned int last = range.y;
    unsigned int split = findSplit(d_morton_codes, first, last);

    uint2 children;
    // set the children, shifting ahead by nleafs - cur_type to account for leaf shifting
    // this factor comes out from resetting 0 = N_leaf,i each time, and then remapping this to
    // an internal node
    children.x = (split == first) ? split : (nleafs - active_types + split);
    children.y = ((split + 1) == last) ? (split + 1) : nleafs - active_types + split + 1;

    uint2 parent_sib;
    parent_sib.x = nleafs + idx;

    // encode the sibling as the right child
    parent_sib.y = children.y << 1;
    parent_sib.y |= 1;

    d_tree_parent_sib[children.x] = parent_sib;

    // encode the sibling as the left child
    parent_sib.y = children.x << 1;
    d_tree_parent_sib[children.y] = parent_sib;

    // root is always number "zero", but only it can set its parent / sibling
    // we mark both of these as the root for traversing, since only the root node
    // will be its own sibling
    if (node_idx == origin)
        {
        parent_sib.x = nleafs + idx;
        parent_sib.y = (nleafs + idx) << 1;

        d_tree_parent_sib[nleafs + idx] = parent_sib;
        }
    }

/*!
 * \param d_tree_parent_sib Parent and sibling for each node in the tree
 * \param d_morton_codes Morton codes for each leaf node
 * \param d_num_per_type Number of particles per type
 * \param ntypes Number of types
 * \param nleafs Number of leafs
 * \param block_size Requested thread block size
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_gen_hierarchy(uint2 *d_tree_parent_sib,
                                    const uint32_t *d_morton_codes,
                                    const unsigned int *d_num_per_type,
                                    const unsigned int ntypes,
                                    const unsigned int nleafs,
                                    const unsigned int ninternal,
                                    const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_gen_hierarchy_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);

    // one thread per internal node
    gpu_nlist_gen_hierarchy_kernel<<<ninternal/run_block_size + 1, run_block_size>>>(d_tree_parent_sib,
                                                                                     d_morton_codes,
                                                                                     d_num_per_type,
                                                                                     ntypes,
                                                                                     nleafs,
                                                                                     ninternal);
    return hipSuccess;
    }

//! Kernel to bubble up enclosing AABBs to internal nodes from leaf nodes
/*!
 * \param d_node_locks Atomic flags identifying when node has been visited
 * \param d_tree_aabbs AABB array for all tree nodes
 * \param d_tree_parent_sib Parent and sibling indexes of each node
 * \param ntypes Number of particle types
 * \param nleafs Number of leaf nodes
 *
 * \b Implementation
 * One thread is called per leaf node. The second thread to reach an internal node processes its two children,
 * which guarantees that no node AABB is prematurely processed. The arrival order at a node is controlled by an atomic
 * thread lock in global memory. This locking could be accelerated by using shared memory whenever a node is being
 * processed by threads in the same block.
 *
 * When processing the node, the thread also walks up the tree to find the "rope" that tells a traverser
 * how to navigate the tree. If a query AABB intersects the current node, then the traverser always moves the the left
 * child of the current node. If the AABB does not intersect, it moves along the "rope" to the next portion of the tree.
 * The "rope" is calculated by walking back up the tree to find the earliest ancestor that is a left child of its
 * parent. The rope then goes to that ancestor's sibling. If the root node is reached, then the rope is set to -1 to
 * indicate traversal should be aborted.
 *
 * This kernel also encodes the left child of a node into the AABB for internal nodes. The thread processing the node
 * checks if it arrived from a left child or right child of the node it is processing, and sets the left child of that
 * parent accordingly. A child is indicated by bit shifting, and setting the first bit to 1.
 */
__global__ void gpu_nlist_bubble_aabbs_kernel(unsigned int *d_node_locks,
                                              Scalar4 *d_tree_aabbs,
                                              const uint2 *d_tree_parent_sib,
                                              const unsigned int ntypes,
                                              const unsigned int nleafs)
    {
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= nleafs)
        return;

    // okay, first we start from the leaf and set my bounding box
    Scalar4 cur_upper = d_tree_aabbs[2*idx];
    Scalar4 cur_lower = d_tree_aabbs[2*idx+1];

    // zero the counters for internal nodes
    cur_upper.w = 0.0f;
    cur_lower.w = 0.0f;

    unsigned int cur_node = idx;
    unsigned int lock_key = 0;
    do
        {
        uint2 cur_parent_sib = d_tree_parent_sib[cur_node];
        unsigned int cur_parent = cur_parent_sib.x;

        // if the current sibling is a right child, then the current node is a left child
        bool cur_is_left = (cur_parent_sib.y & 1);

        unsigned int cur_sibling = cur_parent_sib.y >> 1;

        // first we compute the skip for this node always
        // back track up the tree until you find a left child
        // we have a check in place so that we don't stall on the root node
        uint2 backtrack = cur_parent_sib;
        while (!(backtrack.y & 1) && backtrack.x != (backtrack.y >> 1))
            {
            backtrack = d_tree_parent_sib[backtrack.x];
            }
        // then, the skip is to the sibling of that node, or else to quit
        if (backtrack.y & 1)
            {
            d_tree_aabbs[2*cur_node].w = __int_as_scalar(backtrack.y >> 1);
            }
        else
            {
            d_tree_aabbs[2*cur_node].w = __int_as_scalar(-1);
            }

        // then, we do an atomicAdd on the lock to see if we need to process the parent AABBs
        // check to make sure the parent is bigger than nleafs, or else the node lock always fails
        // so that we terminate the thread
        lock_key = (cur_parent >= nleafs) ? atomicAdd(d_node_locks + cur_parent - nleafs, 1) : 0;

        // process the node
        if (lock_key == 1)
            {
            // compute the max upper bound
            Scalar4 sib_upper = d_tree_aabbs[2*cur_sibling];
            if (sib_upper.x > cur_upper.x) cur_upper.x = sib_upper.x;
            if (sib_upper.y > cur_upper.y) cur_upper.y = sib_upper.y;
            if (sib_upper.z > cur_upper.z) cur_upper.z = sib_upper.z;
            d_tree_aabbs[2*cur_parent] = cur_upper;

            // compute the min lower bound
            Scalar4 sib_lower = d_tree_aabbs[2*cur_sibling+1];
            if (sib_lower.x < cur_lower.x) cur_lower.x = sib_lower.x;
            if (sib_lower.y < cur_lower.y) cur_lower.y = sib_lower.y;
            if (sib_lower.z < cur_lower.z) cur_lower.z = sib_lower.z;

            // this must always be some internal node, so stash the left child of this node here
            unsigned int left_child_masked = ((cur_is_left ? cur_node : cur_sibling) << 1) | 1;
            cur_lower.w = __int_as_scalar( left_child_masked );

            d_tree_aabbs[2*cur_parent+1] = cur_lower;

            // bump the current node one level
            cur_node = cur_parent;
            }
        }
    while (lock_key == 1);

    }

/*!
 * \param d_node_locks Atomic flags identifying when node has been visited
 * \param d_tree_aabbs AABB array for all tree nodes
 * \param d_tree_parent_sib Parent and sibling indexes of each node
 * \param ntypes Number of particle types
 * \param nleafs Number of leaf nodes
 * \param block_size Requested thread block size
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_bubble_aabbs(unsigned int *d_node_locks,
                                   Scalar4 *d_tree_aabbs,
                                   const uint2 *d_tree_parent_sib,
                                   const unsigned int ntypes,
                                   const unsigned int nleafs,
                                   const unsigned int ninternal,
                                   const unsigned int block_size)
    {
    hipMemset(d_node_locks, 0, sizeof(unsigned int)*ninternal);

    gpu_nlist_bubble_aabbs_kernel<<<nleafs/block_size + 1, block_size>>>(d_node_locks,
                                                                         d_tree_aabbs,
                                                                         d_tree_parent_sib,
                                                                         ntypes,
                                                                         nleafs);

    return hipSuccess;
    }

//! Kernel to rearrange particle data into leaf order for faster traversal
/*!
 * \param d_leaf_xyzf Particle xyz coordinates + particle id in leaf order
 * \param d_leaf_db Particle diameter and body id in leaf order
 * \param d_pos Particle positions
 * \param d_diameter Particle diameters
 * \param d_body Particle body ids
 * \param d_map_tree_pid ParticleData indexes corresponding to a leaf particle id
 * \param Ntot Number of particles owned by this rank
 *
 * \b Implementation
 * One thread per particle is called. Writes are coalesced by writing in leaf order, and reading in a scattered way.
 */
__global__ void gpu_nlist_move_particles_kernel(Scalar4 *d_leaf_xyzf,
                                                Scalar2 *d_leaf_db,
                                                const Scalar4 *d_pos,
                                                const Scalar *d_diameter,
                                                const unsigned int *d_body,
                                                const unsigned int *d_map_tree_pid,
                                                const unsigned int Ntot)
    {
    // get thread index
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= Ntot)
        return;

    // read and write particle data
    unsigned int p_idx = d_map_tree_pid[idx];
    Scalar4 pos_i = d_pos[p_idx];
    d_leaf_xyzf[idx] = make_scalar4(pos_i.x, pos_i.y, pos_i.z, __int_as_scalar(p_idx));

    Scalar2 db = make_scalar2(d_diameter[p_idx], __int_as_scalar(d_body[p_idx]));
    d_leaf_db[idx] = db;
    }

/*!
 * \param d_leaf_xyzf Particle xyz coordinates + particle id in leaf order
 * \param d_leaf_db Particle diameter and body id in leaf order
 * \param d_pos Particle positions
 * \param d_diameter Particle diameters
 * \param d_body Particle body ids
 * \param d_map_tree_pid ParticleData indexes corresponding to a leaf particle id
 * \param Ntot Number of particles owned by this rank
 * \param block_size Requested thread block size
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_move_particles(Scalar4 *d_leaf_xyzf,
                                     Scalar2 *d_leaf_db,
                                     const Scalar4 *d_pos,
                                     const Scalar *d_diameter,
                                     const unsigned int *d_body,
                                     const unsigned int *d_map_tree_pid,
                                     const unsigned int Ntot,
                                     const unsigned int block_size)
    {
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_move_particles_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);

    gpu_nlist_move_particles_kernel<<<Ntot/run_block_size + 1, run_block_size>>>(d_leaf_xyzf,
                                                                                 d_leaf_db,
                                                                                 d_pos,
                                                                                 d_diameter,
                                                                                 d_body,
                                                                                 d_map_tree_pid,
                                                                                 Ntot);
    return hipSuccess;
    }


//! Kernel for traversing tree to generate neighbor list
/*!
 * \param d_nlist Neighbor list for writing
 * \param d_n_neigh Number of neighbors per particle
 * \param d_last_updated_pos Records current particle positions
 * \param d_conditions Store overflow condition by type
 * \param d_Nmax Maximum number of neighbors allocated by type
 * \param d_head_list Indexes for writing into neighbor list
 * \param N Number of particles
 * \param nghosts Number of ghost particles
 * \param d_map_tree_pid Map leaf index to local particle index
 * \param d_leaf_offset Offset for reading leaf particles by type
 * \param d_tree_roots Index for tree root by type
 * \param d_tree_aabbs Tree AABBs
 * \param nleafs Total number of leafs
 * \param d_leaf_xyzf Leaf position-id array
 * \param d_leaf_db Leaf diameter-body array
 * \param d_pos Particle positions
 * \param d_image_list Translation vectors to check for traversal
 * \param nimages Number of translation vectors to check
 * \param d_r_cut Cutoff radius by type r_cut(i,j)
 * \param r_buff Buffer around cutoff radius
 * \param max_diam Maximum diameter attained by a particle for diameter shifting
 * \param ntypes Number of particle types
 *
 * \b Implementation
 * One thread is launched per particle, but the threads operate on particles in leaf order rather than ParticleData
 * order in order to minimize divergence within a warp (particles in the same leaf should intersect similar parts of the
 * tree). Each thread iterates on the particle types (trees) and queries on all translation vectors using a stackless
 * search. When the query AABB intersects a node AABB, the node AABB is checked to be an internal node or a leaf node.
 * If an internal node, then the traversal advances to that node's left child. If a leaf node, the leaf particles are
 * tested directly to be included in the neighbor list. The node then advances along that leaf node's rope. If the AABB
 * is not intersected, the traversal advances along the rope. This process proceeds until a rope signals that the
 * traversal is complete.
 */
template<unsigned char flags>
__global__ void gpu_nlist_traverse_tree_kernel(unsigned int *d_nlist,
                                               unsigned int *d_n_neigh,
                                               Scalar4 *d_last_updated_pos,
                                               unsigned int *d_conditions,
                                               const unsigned int *d_Nmax,
                                               const unsigned int *d_head_list,
                                               const unsigned int N,
                                               const unsigned int nghosts,
                                               const unsigned int *d_map_tree_pid,
                                               const unsigned int *d_leaf_offset,
                                               const unsigned int *d_tree_roots,
                                               const Scalar4 *d_tree_aabbs,
                                               const unsigned int nleafs,
                                               const Scalar4 *d_leaf_xyzf,
                                               const Scalar2 *d_leaf_db,
                                               const Scalar4 *d_pos,
                                               const Scalar3 *d_image_list,
                                               const unsigned int nimages,
                                               const Scalar *d_r_cut,
                                               const Scalar r_buff,
                                               const Scalar max_diam,
                                               const unsigned int ntypes)
    {
    bool filter_body = flags & 1;
    bool diameter_shift = flags & 2;

    // cache the r_listsq parameters into shared memory
    const Index2D typpair_idx(ntypes);
    const unsigned int num_typ_parameters = typpair_idx.getNumElements();

    // shared data for per type pair parameters
    extern __shared__ unsigned char s_data[];

    // pointer for the r_listsq data
    Scalar *s_r_list = (Scalar *)(&s_data[0]);
    unsigned int *s_Nmax = (unsigned int *)(&s_data[sizeof(Scalar)*num_typ_parameters]);
    unsigned int *s_leaf_offset = (unsigned int *)(&s_data[sizeof(Scalar)*num_typ_parameters + sizeof(unsigned int)*ntypes]);

    // load in the per type pair r_list
    for (unsigned int cur_offset = 0; cur_offset < num_typ_parameters; cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < num_typ_parameters)
            {
            Scalar r_cut = d_r_cut[cur_offset + threadIdx.x];
            // force the r_list(i,j) to a skippable value if r_cut(i,j) is skippable
            s_r_list[cur_offset + threadIdx.x] = (r_cut > Scalar(0.0)) ? r_cut+r_buff : Scalar(-1.0);
            }

        if (cur_offset + threadIdx.x < ntypes)
            {
            s_Nmax[cur_offset + threadIdx.x] = d_Nmax[cur_offset + threadIdx.x];
            s_leaf_offset[cur_offset + threadIdx.x] = d_leaf_offset[cur_offset + threadIdx.x];
            }
        }
    __syncthreads();


    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // quit now if this thread is processing past the end of the leaf list
    if (idx >= (N+nghosts))
        return;

    // read in the current position
    unsigned int my_pidx = d_map_tree_pid[idx];
    // we only process particles owned by this processor for neighbors
    if (my_pidx >= N)
        return;

    const Scalar4 postype_i = texFetchScalar4(d_pos, pdata_pos_tex, my_pidx);
    const Scalar3 pos_i = make_scalar3(postype_i.x, postype_i.y, postype_i.z);
    const unsigned int type_i = __scalar_as_int(postype_i.w);

    // fetch the diameter and body out of the leaf texture since it's bound anyway
    const Scalar2 db_i = texFetchScalar2(d_leaf_db, leaf_db_tex, idx);
    const Scalar diam_i = db_i.x;
    const unsigned int body_i = __scalar_as_int(db_i.y);

    const unsigned int nlist_head_i = texFetchUint(d_head_list, head_list_tex, my_pidx);

    unsigned int n_neigh_i = 0;
    for (unsigned int cur_pair_type=0; cur_pair_type < ntypes; ++cur_pair_type)
        {
        // Check primary box
        const Scalar r_cut_i = s_r_list[typpair_idx(type_i,cur_pair_type)];

        // Skip this tree type if it is not needed
        if (r_cut_i <= Scalar(0.0))
            continue;

        // stash the r_cutsq before any diameter shifting
        const Scalar r_cutsq_i = r_cut_i*r_cut_i;

        // the rlist to use for the AABB search has to be at least as big as the biggest diameter
        Scalar r_list_i = r_cut_i;
        if (diameter_shift)
            r_list_i += max_diam - Scalar(1.0);

        const unsigned int cur_tree_root = d_tree_roots[cur_pair_type];
        // skip this type if we don't have it
        if (cur_tree_root == NLIST_GPU_INVALID_NODE)
            continue;

        for (unsigned int cur_image = 0; cur_image < nimages; ++cur_image)
            {
            const Scalar3 pos_i_image = pos_i + d_image_list[cur_image];
            const Scalar3 aabb_upper = make_scalar3(pos_i_image.x + r_list_i,
                                                    pos_i_image.y + r_list_i,
                                                    pos_i_image.z + r_list_i);
            const Scalar3 aabb_lower = make_scalar3(pos_i_image.x - r_list_i,
                                                    pos_i_image.y - r_list_i,
                                                    pos_i_image.z - r_list_i);

            // stackless search
            int cur_node_idx = cur_tree_root;
            while (cur_node_idx > -1)
                {
                const Scalar4 upper_rope = texFetchScalar4(d_tree_aabbs, aabb_node_bounds_tex, 2*cur_node_idx);
                const Scalar4 lower_np = texFetchScalar4(d_tree_aabbs, aabb_node_bounds_tex, 2*cur_node_idx+1);

                if (!(aabb_upper.x < lower_np.x
                      || aabb_lower.x > upper_rope.x
                      || aabb_upper.y < lower_np.y
                      || aabb_lower.y > upper_rope.y
                      || aabb_upper.z < lower_np.z
                      || aabb_lower.z > upper_rope.z))
                    {
                    const unsigned int np_child_masked = __scalar_as_int(lower_np.w);

                    if(!(np_child_masked & 1))
                        {
                        // leaf node
                        // all leaves must have at least 1 particle, so we can use this to decide
                        const unsigned int node_head = NLIST_GPU_PARTICLES_PER_LEAF*cur_node_idx - s_leaf_offset[cur_pair_type];
                        const unsigned int n_part = np_child_masked >> 1;
                        for (unsigned int cur_p = node_head; cur_p < node_head + n_part; ++cur_p)
                            {
                            // neighbor j
                            const Scalar4 cur_xyzf = texFetchScalar4(d_leaf_xyzf, leaf_xyzf_tex, cur_p);
                            const Scalar3 pos_j = make_scalar3(cur_xyzf.x, cur_xyzf.y, cur_xyzf.z);
                            const unsigned int j = __scalar_as_int(cur_xyzf.w);

                            const Scalar2 cur_db = texFetchScalar2(d_leaf_db, leaf_db_tex, cur_p);
                            const Scalar diam_j = cur_db.x;
                            const unsigned int body_j = __scalar_as_int(cur_db.y);

                            bool excluded = (my_pidx == j);

                            if (filter_body && body_i != 0xffffffff)
                                excluded = excluded | (body_i == body_j);

                            if (!excluded)
                                {
                                // now we can trim down the actual particles based on diameter
                                // compute the shift for the cutoff if not excluded
                                Scalar sqshift = Scalar(0.0);
                                if (diameter_shift)
                                    {
                                    const Scalar delta = (diam_i + diam_j) * Scalar(0.5) - Scalar(1.0);
                                    // r^2 < (r_list + delta)^2
                                    // r^2 < r_listsq + delta^2 + 2*r_list*delta
                                    sqshift = (delta + Scalar(2.0) * r_cut_i) * delta;
                                    }

                                // compute distance and wrap back into box
                                Scalar3 drij = pos_j - pos_i_image;
                                Scalar dr2 = dot(drij,drij);

                                if (dr2 <= (r_cutsq_i + sqshift))
                                    {
                                    if (n_neigh_i < s_Nmax[type_i])
                                        {
                                        d_nlist[nlist_head_i + n_neigh_i] = j;
                                        }
                                    ++n_neigh_i;
                                    }
                                }
                            }

                        // leaf nodes always move to their rope
                        cur_node_idx = __scalar_as_int(upper_rope.w);
                        }
                    else
                        {
                        // internal node, take left child
                        cur_node_idx = (np_child_masked >> 1);
                        }
                    }
                else
                    {
                    cur_node_idx = __scalar_as_int(upper_rope.w); // no overlap, rope ahead
                    }
                } // end stackless search
            } // end loop over images
        } // end loop over pair types

    // could try reordering by idx instead of pidx, but that seems to not make much difference in microbenchmarking.
    d_n_neigh[my_pidx] = n_neigh_i;
    d_last_updated_pos[my_pidx] = make_scalar4(pos_i.x, pos_i.y, pos_i.z, __scalar_as_int(type_i));

    // update the number of neighbors for this type if allocated memory is exceeded
    if (n_neigh_i >= s_Nmax[type_i])
        atomicMax(&d_conditions[type_i], n_neigh_i);
    }

/*!
 * \param d_nlist Neighbor list for writing
 * \param d_n_neigh Number of neighbors per particle
 * \param d_last_updated_pos Records current particle positions
 * \param d_conditions Store overflow condition by type
 * \param d_Nmax Maximum number of neighbors allocated by type
 * \param d_head_list Indexes for writing into neighbor list
 * \param N Number of particles
 * \param nghosts Number of ghost particles
 * \param d_map_tree_pid Map leaf index to local particle index
 * \param d_leaf_offset Offset for reading leaf particles by type
 * \param d_tree_roots Index for tree root by type
 * \param d_tree_aabbs Tree AABBs
 * \param nleafs Total number of leafs
 * \param d_leaf_xyzf Leaf position-id array
 * \param d_leaf_db Leaf diameter-body array
 * \param d_pos Particle positions
 * \param d_image_list Translation vectors to check for traversal
 * \param nimages Number of translation vectors to check
 * \param d_r_cut Cutoff radius by type r_cut(i,j)
 * \param r_buff Buffer around cutoff radius
 * \param max_diam Maximum diameter attained by a particle for diameter shifting
 * \param ntypes Number of particle types
 * \param filter_body True if body filtering is enabled
 * \param diameter_shift True if rcut(i,j) should be shifted by the particle diameters
 * \param compute_capability Compute capability of the GPU (in 20, 30, 35 format)
 * \param block_size Requested thread block size
 *
 * \returns hipSuccess on completion
 * \returns hipError_t on failure to texture bind
 *
 * \note Kernel calls are templated on body filtering and diameter shifting for optimization.
 * \note One thread is called for all leaf particles. Some of these threads will die because they correspond to ghost
 *       particles not owned by the rank. Because the leaf particles are sorted, there is no easy way to skip these
 *       particles, and this inefficiency is assumed to be relatively small.
 */
hipError_t gpu_nlist_traverse_tree(unsigned int *d_nlist,
                                    unsigned int *d_n_neigh,
                                    Scalar4 *d_last_updated_pos,
                                    unsigned int *d_conditions,
                                    const unsigned int *d_Nmax,
                                    const unsigned int *d_head_list,
                                    const unsigned int N,
                                    const unsigned int nghosts,
                                    const unsigned int *d_map_tree_pid,
                                    const unsigned int *d_leaf_offset,
                                    const unsigned int *d_tree_roots,
                                    const Scalar4 *d_tree_aabbs,
                                    const unsigned int nleafs,
                                    const unsigned int ninternal,
                                    const unsigned int nnodes,
                                    const Scalar4 *d_leaf_xyzf,
                                    const Scalar2 *d_leaf_db,
                                    const Scalar4 *d_pos,
                                    const Scalar3 *d_image_list,
                                    const unsigned int nimages,
                                    const Scalar *d_r_cut,
                                    const Scalar r_buff,
                                    const Scalar max_diam,
                                    const unsigned int ntypes,
                                    bool filter_body,
                                    bool diameter_shift,
                                    const unsigned int compute_capability,
                                    const unsigned int block_size)
    {
    // shared memory = r_list + Nmax
    Index2D typpair_idx(ntypes);
    unsigned int shared_size = sizeof(Scalar)*typpair_idx.getNumElements() + 2*sizeof(unsigned int)*ntypes;

    // bind the neighborlist texture
    if (compute_capability < 35)
        {
        pdata_pos_tex.normalized = false;
        pdata_pos_tex.filterMode = hipFilterModePoint;
        hipError_t error = hipBindTexture(0, pdata_pos_tex, d_pos, sizeof(Scalar4)*(N+nghosts));
        if (error != hipSuccess)
            return error;

        leaf_xyzf_tex.normalized = false;
        leaf_xyzf_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, leaf_xyzf_tex, d_leaf_xyzf, sizeof(Scalar4)*(N+nghosts));
        if (error != hipSuccess)
            return error;

        leaf_db_tex.normalized = false;
        leaf_db_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, leaf_db_tex, d_leaf_db, sizeof(Scalar2)*(N+nghosts));
        if (error != hipSuccess)
            return error;

        aabb_node_bounds_tex.normalized = false;
        aabb_node_bounds_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, aabb_node_bounds_tex, d_tree_aabbs, sizeof(Scalar4)*2*nnodes);
        if (error != hipSuccess)
            return error;

        head_list_tex.normalized = false;
        head_list_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, head_list_tex, d_head_list, sizeof(unsigned int)*N);
        if (error != hipSuccess)
            return error;
        }

    if (!filter_body && !diameter_shift)
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_nlist_traverse_tree_kernel<0>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        int run_block_size = min(block_size,max_block_size);
        int nblocks = (N+nghosts)/run_block_size + 1;
        gpu_nlist_traverse_tree_kernel<0><<<nblocks, run_block_size, shared_size>>>(d_nlist,
                                                                                    d_n_neigh,
                                                                                    d_last_updated_pos,
                                                                                    d_conditions,
                                                                                    d_Nmax,
                                                                                    d_head_list,
                                                                                    N,
                                                                                    nghosts,
                                                                                    d_map_tree_pid,
                                                                                    d_leaf_offset,
                                                                                    d_tree_roots,
                                                                                    d_tree_aabbs,
                                                                                    nleafs,
                                                                                    d_leaf_xyzf,
                                                                                    d_leaf_db,
                                                                                    d_pos,
                                                                                    d_image_list,
                                                                                    nimages,
                                                                                    d_r_cut,
                                                                                    r_buff,
                                                                                    max_diam,
                                                                                    ntypes);
        }
    else if (filter_body && !diameter_shift)
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_nlist_traverse_tree_kernel<1>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        int run_block_size = min(block_size,max_block_size);
        int nblocks = (N+nghosts)/run_block_size + 1;
        gpu_nlist_traverse_tree_kernel<1><<<nblocks, run_block_size, shared_size>>>(d_nlist,
                                                                                    d_n_neigh,
                                                                                    d_last_updated_pos,
                                                                                    d_conditions,
                                                                                    d_Nmax,
                                                                                    d_head_list,
                                                                                    N,
                                                                                    nghosts,
                                                                                    d_map_tree_pid,
                                                                                    d_leaf_offset,
                                                                                    d_tree_roots,
                                                                                    d_tree_aabbs,
                                                                                    nleafs,
                                                                                    d_leaf_xyzf,
                                                                                    d_leaf_db,
                                                                                    d_pos,
                                                                                    d_image_list,
                                                                                    nimages,
                                                                                    d_r_cut,
                                                                                    r_buff,
                                                                                    max_diam,
                                                                                    ntypes);
        }
    else if (!filter_body && diameter_shift)
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_nlist_traverse_tree_kernel<2>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        int run_block_size = min(block_size,max_block_size);
        int nblocks = (N+nghosts)/run_block_size + 1;
        gpu_nlist_traverse_tree_kernel<2><<<nblocks, run_block_size, shared_size>>>(d_nlist,
                                                                                    d_n_neigh,
                                                                                    d_last_updated_pos,
                                                                                    d_conditions,
                                                                                    d_Nmax,
                                                                                    d_head_list,
                                                                                    N,
                                                                                    nghosts,
                                                                                    d_map_tree_pid,
                                                                                    d_leaf_offset,
                                                                                    d_tree_roots,
                                                                                    d_tree_aabbs,
                                                                                    nleafs,
                                                                                    d_leaf_xyzf,
                                                                                    d_leaf_db,
                                                                                    d_pos,
                                                                                    d_image_list,
                                                                                    nimages,
                                                                                    d_r_cut,
                                                                                    r_buff,
                                                                                    max_diam,
                                                                                    ntypes);
        }
    else if (filter_body && diameter_shift)
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_nlist_traverse_tree_kernel<3>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        int run_block_size = min(block_size,max_block_size);
        int nblocks = (N+nghosts)/run_block_size + 1;
        gpu_nlist_traverse_tree_kernel<3><<<nblocks, run_block_size, shared_size>>>(d_nlist,
                                                                                    d_n_neigh,
                                                                                    d_last_updated_pos,
                                                                                    d_conditions,
                                                                                    d_Nmax,
                                                                                    d_head_list,
                                                                                    N,
                                                                                    nghosts,
                                                                                    d_map_tree_pid,
                                                                                    d_leaf_offset,
                                                                                    d_tree_roots,
                                                                                    d_tree_aabbs,
                                                                                    nleafs,
                                                                                    d_leaf_xyzf,
                                                                                    d_leaf_db,
                                                                                    d_pos,
                                                                                    d_image_list,
                                                                                    nimages,
                                                                                    d_r_cut,
                                                                                    r_buff,
                                                                                    max_diam,
                                                                                    ntypes);
        }

    // unbind the textures
    if (compute_capability < 35)
        {
        hipError_t error = hipUnbindTexture(pdata_pos_tex);
        if (error != hipSuccess)
            return error;

        error = hipUnbindTexture(leaf_xyzf_tex);
        if (error != hipSuccess)
            return error;

        error = hipUnbindTexture(leaf_db_tex);
        if (error != hipSuccess)
            return error;

        error = hipUnbindTexture(aabb_node_bounds_tex);
        if (error != hipSuccess)
            return error;

        error = hipUnbindTexture(head_list_tex);
        if (error != hipSuccess)
            return error;
        }

    return hipSuccess;
    }

//! Kernel to find divisons between particle types in sorted order
/*!
 * \param d_type_head Index to first type in leaf ordered particles by type
 * \param d_pos Particle positions
 * \param d_map_tree_pid ParticleData indexes corresponding to a leaf particle id
 * \param N Total number of particles on rank (including ghosts)
 *
 * The starting index for each type of particles is the first particle where the left neighbor is not of the same type.
 */
__global__ void gpu_nlist_get_divisions_kernel(unsigned int *d_type_head,
                                               const Scalar4 *d_pos,
                                               const unsigned int *d_map_tree_pid,
                                               const unsigned int N)
    {
    // compute the particle index this thread operates on
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // one thread per particle
    if (idx >= N)
        return;

    const unsigned int cur_pidx = d_map_tree_pid[idx];
    // get type of the current particle
    const Scalar4 cur_postype = d_pos[cur_pidx];
    const unsigned int cur_type = __scalar_as_int(cur_postype.w);

    // all particles except for the first one should look left
    if (idx > 0)
        {
        const unsigned int left_pidx = d_map_tree_pid[idx - 1];

        // get type of the particle to my left
        const Scalar4 left_postype = d_pos[left_pidx];
        const unsigned int left_type = __scalar_as_int(left_postype.w);

        // if the left has a different type, then this is a type boundary, and the type starts at the current thread index
        if (left_type != cur_type)
            {
            d_type_head[cur_type] = idx + 1; // offset the index +1 so that we can use 0 to mean "none of this found"
            }
        }
    else // the first particle just sets its type to be 1
        {
        d_type_head[cur_type] = 1;
        }
    }

/*!
 * \param d_type_head Index to first type in leaf ordered particles by type
 * \param d_num_per_type Number of particles per type
 * \param d_leaf_offset Offset for reading particles out of leaf order
 * \param d_tree_roots Root node of each tree
 * \param d_pos Particles positions
 * \param d_map_tree_pid ParticleData indexes corresponding to a leaf particle id
 * \param N Total number of particles on rank (including ghosts)
 * \param ntypes Number of types
 * \param block_size Requested thread block size
 *
 * \returns hipSuccess on completion
 */
hipError_t gpu_nlist_init_count(unsigned int *d_type_head,
                                 const Scalar4 *d_pos,
                                 const unsigned int *d_map_tree_pid,
                                 const unsigned int N,
                                 const unsigned int ntypes,
                                 const unsigned int block_size)
    {
    // apply the scan
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((const void *))gpu_nlist_get_divisions_kernel);
        max_block_size = attr.maxThreadsPerBlock;
        }

    int run_block_size = min(block_size,max_block_size);

    // zero out the head list
    hipMemset(d_type_head, 0, sizeof(unsigned int)*ntypes);

    // get the head list divisions
    gpu_nlist_get_divisions_kernel<<<N/run_block_size + 1, run_block_size>>>(d_type_head, d_pos, d_map_tree_pid, N);

    return hipSuccess;
    }

#undef MORTON_CODE_BITS
#undef MORTON_TYPE_MASK_64
#undef MORTON_CODE_N_BINS
