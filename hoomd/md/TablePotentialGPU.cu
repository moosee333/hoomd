#include "hip/hip_runtime.h"
// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: joaander

#include "TablePotentialGPU.cuh"
#include "hoomd/TextureTools.h"

#include "hoomd/Index1D.h"

#include <assert.h>

/*! \file TablePotentialGPU.cu
    \brief Defines GPU kernel code for calculating the table pair forces. Used by TablePotentialGPU.
*/

//! Texture for reading particle positions
scalar4_tex_t pdata_pos_tex;

//! Texture for reading the neighborlist
texture<unsigned int, 1, hipReadModeElementType> nlist_tex;

//! Texture for reading table values
scalar2_tex_t tables_tex;

/*!  This kernel is called to calculate the table pair forces on all N particles

    \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch Pitch of 2D virial array
    \param N number of particles in system
    \param d_pos device array of particle positions
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param d_head_list Indexer for reading \a d_nlist
    \param d_params Parameters for each table associated with a type pair
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table

    See TablePotential for information on the memory layout.

    \tparam use_gmem_nlist When non-zero, the neighbor list is read out of global memory. When zero, textures or __ldg
                           is used depending on architecture.

    \b Details:
    * Table entries are read from tables_tex. Note that currently this is bound to a 1D memory region. Performance tests
      at a later date may result in this changing.
*/
template<unsigned char use_gmem_nlist>
__global__ void gpu_compute_table_forces_kernel(Scalar4* d_force,
                                                Scalar* d_virial,
                                                const unsigned virial_pitch,
                                                const unsigned int N,
                                                const Scalar4 *d_pos,
                                                const BoxDim box,
                                                const unsigned int *d_n_neigh,
                                                const unsigned int *d_nlist,
                                                const unsigned int *d_head_list,
                                                const Scalar2 *d_tables,
                                                const Scalar4 *d_params,
                                                const unsigned int ntypes,
                                                const unsigned int table_width)
    {
    // index calculation helpers
    Index2DUpperTriangular table_index(ntypes);
    Index2D table_value(table_width);

    // read in params for easy and fast access in the kernel
    extern __shared__ Scalar4 s_params[];
    for (unsigned int cur_offset = 0; cur_offset < table_index.getNumElements(); cur_offset += blockDim.x)
        {
        if (cur_offset + threadIdx.x < table_index.getNumElements())
            s_params[cur_offset + threadIdx.x] = d_params[cur_offset + threadIdx.x];
        }
    __syncthreads();

    // start by identifying which particle we are to handle
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    // load in the length of the list
    unsigned int n_neigh = d_n_neigh[idx];
    const unsigned int head_idx = d_head_list[idx];

    // read in the position of our particle. Texture reads of Scalar4's are faster than global reads on compute 1.0 hardware
    Scalar4 postype = texFetchScalar4(d_pos, pdata_pos_tex, idx);
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    unsigned int typei = __scalar_as_int(postype.w);

    // initialize the force to 0
    Scalar4 force = make_scalar4(Scalar(0.0), Scalar(0.0), Scalar(0.0), Scalar(0.0));
    Scalar virialxx = Scalar(0.0);
    Scalar virialxy = Scalar(0.0);
    Scalar virialxz = Scalar(0.0);
    Scalar virialyy = Scalar(0.0);
    Scalar virialyz = Scalar(0.0);
    Scalar virialzz = Scalar(0.0);

    // prefetch neighbor index
    unsigned int cur_neigh = 0;
    unsigned int next_neigh(0);
    if (use_gmem_nlist)
        {
        next_neigh = d_nlist[head_idx];
        }
    else
        {
        next_neigh = texFetchUint(d_nlist, nlist_tex, head_idx);
        }

    // loop over neighbors
    for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
        {
        // read the current neighbor index
        // prefetch the next value and set the current one
        cur_neigh = next_neigh;
        if (use_gmem_nlist)
            {
            next_neigh = d_nlist[head_idx + neigh_idx + 1];
            }
        else
            {
            next_neigh = texFetchUint(d_nlist, nlist_tex, head_idx + neigh_idx+1);
            }

        // get the neighbor's position
        Scalar4 neigh_postype = texFetchScalar4(d_pos, pdata_pos_tex, cur_neigh);
        Scalar3 neigh_pos = make_scalar3(neigh_postype.x, neigh_postype.y, neigh_postype.z);

        // calculate dr (with periodic boundary conditions)
        Scalar3 dx = pos - neigh_pos;

        // apply periodic boundary conditions
        dx = box.minImage(dx);

        // access needed parameters
        unsigned int typej = __scalar_as_int(neigh_postype.w);
        unsigned int cur_table_index = table_index(typei, typej);
        Scalar4 params = s_params[cur_table_index];
        Scalar rmin = params.x;
        Scalar rmax = params.y;
        Scalar delta_r = params.z;

        // calculate r
        Scalar rsq = dot(dx, dx);
        Scalar r = sqrtf(rsq);

        if (r < rmax && r >= rmin)
            {
            // precomputed term
            Scalar value_f = (r - rmin) / delta_r;

            // compute index into the table and read in values
            unsigned int value_i = floor(value_f);
            Scalar2 VF0 = texFetchScalar2(d_tables, tables_tex, table_value(value_i, cur_table_index));
            Scalar2 VF1 = texFetchScalar2(d_tables, tables_tex, table_value(value_i+1, cur_table_index));

            // unpack the data
            Scalar V0 = VF0.x;
            Scalar V1 = VF1.x;
            Scalar F0 = VF0.y;
            Scalar F1 = VF1.y;

            // compute the linear interpolation coefficient
            Scalar f = value_f - Scalar(value_i);

            // interpolate to get V and F;
            Scalar V = V0 + f * (V1 - V0);
            Scalar F = F0 + f * (F1 - F0);

            // convert to standard variables used by the other pair computes in HOOMD-blue
            Scalar forcemag_divr = Scalar(0.0);
            if (r > Scalar(0.0))
                forcemag_divr = F / r;
            Scalar pair_eng = V;
            // calculate the virial
            Scalar force_div2r = Scalar(0.5) * forcemag_divr;
            virialxx +=  dx.x * dx.x * force_div2r;
            virialxy +=  dx.x * dx.y * force_div2r;
            virialxz +=  dx.x * dx.z * force_div2r;
            virialyy +=  dx.y * dx.y * force_div2r;
            virialyz +=  dx.y * dx.z * force_div2r;
            virialzz +=  dx.z * dx.z * force_div2r;

            // add up the force vector components (FLOPS: 7)
            force.x += dx.x * forcemag_divr;
            force.y += dx.y * forcemag_divr;
            force.z += dx.z * forcemag_divr;
            force.w += pair_eng;
            }
        }

    // potential energy per particle must be halved
    force.w *= Scalar(0.5);
    // now that the force calculation is complete, write out the result
    d_force[idx] = force;
    d_virial[0*virial_pitch+idx] = virialxx;
    d_virial[1*virial_pitch+idx] = virialxy;
    d_virial[2*virial_pitch+idx] = virialxz;
    d_virial[3*virial_pitch+idx] = virialyy;
    d_virial[4*virial_pitch+idx] = virialyz;
    d_virial[5*virial_pitch+idx] = virialzz;
    }

/*! \param d_force Device memory to write computed forces
    \param d_virial Device memory to write computed virials
    \param virial_pitch pitch of 2D virial array
    \param N number of particles
    \param n_ghost number of ghost particles
    \param d_pos particle positions on the device
    \param box Box dimensions used to implement periodic boundary conditions
    \param d_n_neigh Device memory array listing the number of neighbors for each particle
    \param d_nlist Device memory array containing the neighbor list contents
    \param d_head_list Indexer for reading \a d_nlist
    \param d_tables Tables of the potential and force
    \param d_params Parameters for each table associated with a type pair
    \param size_nlist Total length of the neighborlist
    \param ntypes Number of particle types in the system
    \param table_width Number of points in each table
    \param block_size Block size at which to run the kernel
    \param compute_capability Compute capability of the device (200, 300, 350)
    \param max_tex1d_width Maximum width of a linear 1d texture

    \note This is just a kernel driver. See gpu_compute_table_forces_kernel for full documentation.
*/
hipError_t gpu_compute_table_forces(Scalar4* d_force,
                                     Scalar* d_virial,
                                     const unsigned int virial_pitch,
                                     const unsigned int N,
                                     const unsigned int n_ghost,
                                     const Scalar4 *d_pos,
                                     const BoxDim& box,
                                     const unsigned int *d_n_neigh,
                                     const unsigned int *d_nlist,
                                     const unsigned int *d_head_list,
                                     const Scalar2 *d_tables,
                                     const Scalar4 *d_params,
                                     const unsigned int size_nlist,
                                     const unsigned int ntypes,
                                     const unsigned int table_width,
                                     const unsigned int block_size,
                                     const unsigned int compute_capability,
                                     const unsigned int max_tex1d_width)
    {
    assert(d_params);
    assert(d_tables);
    assert(ntypes > 0);
    assert(table_width > 1);

    // index calculation helper
    Index2DUpperTriangular table_index(ntypes);

    // texture bind
    if (compute_capability < 350)
        {
        // bind the pdata position texture
        pdata_pos_tex.normalized = false;
        pdata_pos_tex.filterMode = hipFilterModePoint;
        hipError_t error = hipBindTexture(0, pdata_pos_tex, d_pos, sizeof(Scalar4) * (N+n_ghost));
        if (error != hipSuccess)
            return error;

        if (size_nlist <= max_tex1d_width)
            {
            nlist_tex.normalized = false;
            nlist_tex.filterMode = hipFilterModePoint;
            error = hipBindTexture(0, nlist_tex, d_nlist, sizeof(unsigned int)*size_nlist);
            if (error != hipSuccess)
                return error;
            }

        // bind the tables texture
        tables_tex.normalized = false;
        tables_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, tables_tex, d_tables, sizeof(Scalar2) * table_width * table_index.getNumElements());
        if (error != hipSuccess)
            return error;
        }

    if (compute_capability < 350 && size_nlist > max_tex1d_width)
        { // use global memory when the neighbor list must be texture bound, but exceeds the max size of a texture
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_compute_table_forces_kernel<1>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);

        // setup the grid to run the kernel
        dim3 grid( N / run_block_size + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        gpu_compute_table_forces_kernel<1><<< grid, threads, sizeof(Scalar4)*table_index.getNumElements() >>>(d_force,
                                                                                                           d_virial,
                                                                                                           virial_pitch,
                                                                                                           N,
                                                                                                           d_pos,
                                                                                                           box,
                                                                                                           d_n_neigh,
                                                                                                           d_nlist,
                                                                                                           d_head_list,
                                                                                                           d_tables,
                                                                                                           d_params,
                                                                                                           ntypes,
                                                                                                           table_width);
        }
    else
        {
        static unsigned int max_block_size = UINT_MAX;
        if (max_block_size == UINT_MAX)
            {
            hipFuncAttributes attr;
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_compute_table_forces_kernel<0>));
            max_block_size = attr.maxThreadsPerBlock;
            }

        unsigned int run_block_size = min(block_size, max_block_size);

        // index calculation helper
        Index2DUpperTriangular table_index(ntypes);

        // setup the grid to run the kernel
        dim3 grid( N / run_block_size + 1, 1, 1);
        dim3 threads(run_block_size, 1, 1);

        gpu_compute_table_forces_kernel<0><<< grid, threads, sizeof(Scalar4)*table_index.getNumElements() >>>(d_force,
                                                                                                           d_virial,
                                                                                                           virial_pitch,
                                                                                                           N,
                                                                                                           d_pos,
                                                                                                           box,
                                                                                                           d_n_neigh,
                                                                                                           d_nlist,
                                                                                                           d_head_list,
                                                                                                           d_tables,
                                                                                                           d_params,
                                                                                                           ntypes,
                                                                                                           table_width);
        }

    return hipSuccess;
    }
// vim:syntax=cpp
