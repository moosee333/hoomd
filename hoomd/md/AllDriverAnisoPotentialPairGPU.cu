// Copyright (c) 2009-2017 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: jglaser

#include "AllDriverAnisoPotentialPairGPU.cuh"

/*! \file AllDriverAnisoPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of anisotropic pair forces on the GPU
*/

hipError_t gpu_compute_pair_aniso_forces_gb(const a_pair_args_t& pair_args,
            const EvaluatorPairGB::param_type* d_param)
    {
    return gpu_compute_pair_aniso_forces<EvaluatorPairGB>(pair_args, d_param);
    }

hipError_t gpu_compute_pair_aniso_forces_dipole(const a_pair_args_t& pair_args,
            const EvaluatorPairDipole::param_type* d_param)
    {
    return gpu_compute_pair_aniso_forces<EvaluatorPairDipole>(pair_args, d_param);
    }
